#include "hip/hip_runtime.h"
//////////////////////////////////////////////////////////////////////////////////////
//
// (C) Daniel Strano and the Qrack contributors 2017-2023. All rights reserved.
//
// This is a multithreaded, universal quantum register simulation, allowing
// (nonphysical) register cloning and direct measurement of probability and
// phase, to leverage what advantages classical emulation of qubits can have.
//
// Licensed under the GNU Lesser General Public License V3.
// See LICENSE.md in the project root or https://www.gnu.org/licenses/lgpl-3.0.en.html
// for details.

#include "common/cuda_kernels.cuh"
#include "qengine_cuda.hpp"

#include <algorithm>
#include <thread>

namespace Qrack {

// Mask definition for Apply2x2()
#define APPLY2X2_DEFAULT 0x00
#define APPLY2X2_NORM 0x01
#define APPLY2X2_SINGLE 0x02
#define APPLY2X2_DOUBLE 0x04
#define APPLY2X2_WIDE 0x08
#define APPLY2X2_X 0x10
#define APPLY2X2_Z 0x20
#define APPLY2X2_PHASE 0x40
#define APPLY2X2_INVERT 0x80

// These are commonly used emplace patterns, for OpenCL buffer I/O.
#define DISPATCH_BLOCK_WRITE(buff, offset, length, array)                                                              \
    clFinish();                                                                                                        \
    tryCuda("Failed to write buffer", [&] {                                                                            \
        return hipMemcpy((void*)((complex*)(buff.get()) + offset), (void*)(array), length, hipMemcpyHostToDevice);   \
    });

#define DISPATCH_TEMP_WRITE(buff, size, array)                                                                         \
    tryCuda("Failed to write buffer", [&] {                                                                            \
        return hipMemcpyAsync(buff.get(), array, size, hipMemcpyHostToDevice, device_context->params_queue);         \
    });

#define DISPATCH_WRITE(buff, size, array)                                                                              \
    tryCuda("Failed to enqueue buffer write", [&] {                                                                    \
        return hipMemcpyAsync(                                                                                        \
            buff.get(), (void*)(array), size, hipMemcpyHostToDevice, device_context->params_queue);                   \
    });

#define DISPATCH_BLOCK_READ(buff, offset, length, array)                                                               \
    clFinish();                                                                                                        \
    tryCuda("Failed to read buffer", [&] {                                                                             \
        return hipMemcpy((void*)(array), (void*)((complex*)(buff.get()) + offset), length, hipMemcpyDeviceToHost);   \
    });

#define WAIT_REAL1_SUM(buff, size, array, sumPtr)                                                                      \
    clFinish();                                                                                                        \
    tryCuda("Failed to enqueue buffer read",                                                                           \
        [&] { return hipMemcpy((void*)((array).get()), buff.get(), sizeof(real1) * size, hipMemcpyDeviceToHost); }); \
    *(sumPtr) = ParSum(array.get(), size);

#define CHECK_ZERO_SKIP()                                                                                              \
    if (!stateBuffer) {                                                                                                \
        return;                                                                                                        \
    }

#define GRID_SIZE (item.workItemCount / item.localGroupSize)
// clang-format off
#define CUDA_KERNEL_2(fn, t0, t1) fn<<<GRID_SIZE, item.localGroupSize, item.localBuffSize, device_context->queue>>>((t0*)(args[0].get()), (t1*)(args[1].get()))
#define CUDA_KERNEL_3(fn, t0, t1, t2) fn<<<GRID_SIZE, item.localGroupSize, item.localBuffSize, device_context->queue>>>((t0*)(args[0].get()), (t1*)(args[1].get()), (t2*)(args[2].get()))
#define CUDA_KERNEL_4(fn, t0, t1, t2, t3) fn<<<GRID_SIZE, item.localGroupSize, item.localBuffSize, device_context->queue>>>((t0*)(args[0].get()), (t1*)(args[1].get()), (t2*)(args[2].get()), (t3*)(args[3].get()))
#define CUDA_KERNEL_5(fn, t0, t1, t2, t3, t4) fn<<<GRID_SIZE, item.localGroupSize, item.localBuffSize, device_context->queue>>>((t0*)(args[0].get()), (t1*)(args[1].get()), (t2*)(args[2].get()), (t3*)(args[3].get()), (t4*)(args[4].get()))
#define CUDA_KERNEL_6(fn, t0, t1, t2, t3, t4, t5) fn<<<GRID_SIZE, item.localGroupSize, item.localBuffSize, device_context->queue>>>((t0*)(args[0].get()), (t1*)(args[1].get()), (t2*)(args[2].get()), (t3*)(args[3].get()), (t4*)(args[4].get()), (t5*)(args[5].get()))
// clang-format on

QEngineCUDA::QEngineCUDA(bitLenInt qBitCount, bitCapInt initState, qrack_rand_gen_ptr rgp, complex phaseFac,
    bool doNorm, bool randomGlobalPhase, bool useHostMem, int64_t devID, bool useHardwareRNG, bool ignored,
    real1_f norm_thresh, std::vector<int64_t> devList, bitLenInt qubitThreshold, real1_f sep_thresh)
    : QEngine(qBitCount, rgp, doNorm, randomGlobalPhase, useHostMem, useHardwareRNG, norm_thresh)
    , didInit(false)
    , unlockHostMem(false)
    , nrmGroupSize(0U)
    , totalOclAllocSize(0U)
    , deviceID(devID)
    , nrmArray(new real1[0], [](real1* r) { delete[] r; })
{
    InitOCL(devID);
    clFinish();
    if (qubitCount) {
        SetPermutation(initState, phaseFac);
    } else {
        ZeroAmplitudes();
    }
}

void QEngineCUDA::FreeAll()
{
    ZeroAmplitudes();

    nrmBuffer = NULL;
    nrmArray = NULL;

    SubtractAlloc(totalOclAllocSize);
}

void QEngineCUDA::ZeroAmplitudes()
{
    clDump();
    runningNorm = ZERO_R1;

    if (!stateBuffer) {
        return;
    }

    ResetStateBuffer(NULL);
    FreeStateVec();

    SubtractAlloc(sizeof(complex) * maxQPowerOcl);
}

void QEngineCUDA::CopyStateVec(QEnginePtr src)
{
    if (qubitCount != src->GetQubitCount()) {
        throw std::invalid_argument("QEngineCUDA::CopyStateVec argument size differs from this!");
    }

    if (src->IsZeroAmplitude()) {
        ZeroAmplitudes();
        return;
    }

    if (stateBuffer) {
        clDump();
    } else {
        ReinitBuffer();
    }

    LockSync(CL_MAP_WRITE);
    src->GetQuantumState(stateVec.get());
    UnlockSync();

    runningNorm = src->GetRunningNorm();
}

void QEngineCUDA::GetAmplitudePage(complex* pagePtr, bitCapIntOcl offset, bitCapIntOcl length)
{
    if (isBadPermRange(offset, length, maxQPowerOcl)) {
        throw std::invalid_argument("QEngineCUDA::GetAmplitudePage range is out-of-bounds!");
    }

    if (!stateBuffer) {
        std::fill(pagePtr, pagePtr + length, ZERO_CMPLX);
        return;
    }

    DISPATCH_BLOCK_READ(stateBuffer, offset, sizeof(complex) * length, pagePtr);
}

void QEngineCUDA::SetAmplitudePage(const complex* pagePtr, bitCapIntOcl offset, bitCapIntOcl length)
{
    if (isBadPermRange(offset, length, maxQPowerOcl)) {
        throw std::invalid_argument("QEngineCUDA::SetAmplitudePage range is out-of-bounds!");
    }

    if (!stateBuffer) {
        ReinitBuffer();
        if (length != maxQPowerOcl) {
            ClearBuffer(stateBuffer, 0U, maxQPowerOcl);
        }
    }

    DISPATCH_BLOCK_WRITE(stateBuffer, offset, sizeof(complex) * length, pagePtr);

    runningNorm = REAL1_DEFAULT_ARG;
}

void QEngineCUDA::SetAmplitudePage(
    QEnginePtr pageEnginePtr, bitCapIntOcl srcOffset, bitCapIntOcl dstOffset, bitCapIntOcl length)
{
    if (isBadPermRange(dstOffset, length, maxQPowerOcl)) {
        throw std::invalid_argument("QEngineCUDA::SetAmplitudePage source range is out-of-bounds!");
    }

    QEngineCUDAPtr pageEngineOclPtr = std::dynamic_pointer_cast<QEngineCUDA>(pageEnginePtr);

    if (isBadPermRange(srcOffset, length, pageEngineOclPtr->maxQPowerOcl)) {
        throw std::invalid_argument("QEngineCUDA::SetAmplitudePage source range is out-of-bounds!");
    }

    BufferPtr oStateBuffer = pageEngineOclPtr->stateBuffer;

    if (!stateBuffer && !oStateBuffer) {
        return;
    }

    if (!oStateBuffer) {
        if (length == maxQPowerOcl) {
            ZeroAmplitudes();
        } else {
            ClearBuffer(stateBuffer, dstOffset, length);
            runningNorm = REAL1_DEFAULT_ARG;
        }

        return;
    }

    if (!stateBuffer) {
        ReinitBuffer();
        ClearBuffer(stateBuffer, 0U, maxQPowerOcl);
    }

    pageEngineOclPtr->clFinish();

    tryCuda("Failed to enqueue buffer copy", [&] {
        return hipMemcpy(oStateBuffer.get(), stateBuffer.get(), sizeof(complex) * srcOffset, hipMemcpyDeviceToDevice);
    });

    runningNorm = REAL1_DEFAULT_ARG;
}

void QEngineCUDA::ShuffleBuffers(QEnginePtr engine)
{
    if (qubitCount != engine->GetQubitCount()) {
        throw std::invalid_argument("QEngineCUDA::ShuffleBuffers argument size differs from this!");
    }

    QEngineCUDAPtr engineOcl = std::dynamic_pointer_cast<QEngineCUDA>(engine);

    if (!stateBuffer && !(engineOcl->stateBuffer)) {
        return;
    }

    if (!stateBuffer) {
        ReinitBuffer();
        ClearBuffer(stateBuffer, 0U, maxQPowerOcl);
    }

    if (!(engineOcl->stateBuffer)) {
        engineOcl->ReinitBuffer();
        engineOcl->ClearBuffer(engineOcl->stateBuffer, 0U, engineOcl->maxQPowerOcl);
    }

    const bitCapIntOcl halfMaxQPower = maxQPowerOcl >> 1U;

    const bitCapIntOcl bciArgs[BCI_ARG_LEN]{ halfMaxQPower, 0U, 0U, 0U, 0U, 0U, 0U, 0U, 0U, 0U };

    PoolItemPtr poolItem = GetFreePoolItem();

    DISPATCH_TEMP_WRITE(poolItem->ulongBuffer, sizeof(bitCapIntOcl), bciArgs);

    const size_t ngc = FixWorkItemCount(halfMaxQPower, nrmGroupCount);
    const size_t ngs = FixGroupSize(ngc, nrmGroupSize);

    engineOcl->clFinish();
    WaitCall(OCL_API_SHUFFLEBUFFERS, ngc, ngs, { stateBuffer, engineOcl->stateBuffer, poolItem->ulongBuffer });

    runningNorm = REAL1_DEFAULT_ARG;
    engineOcl->runningNorm = REAL1_DEFAULT_ARG;
}

void QEngineCUDA::LockSync(cl_map_flags flags)
{
    lockSyncFlags = flags;

    if (stateVec) {
        unlockHostMem = true;
        clFinish();
        tryCuda("Failed to map buffer", [&] {
            return hipMemcpy(
                (void*)(stateVec.get()), stateBuffer.get(), sizeof(complex) * maxQPowerOcl, hipMemcpyDeviceToHost);
        });
    } else {
        unlockHostMem = false;
        stateVec = AllocStateVec(maxQPowerOcl, true);
        if (lockSyncFlags & CL_MAP_READ) {
            DISPATCH_BLOCK_READ(stateBuffer, 0U, sizeof(complex) * maxQPowerOcl, stateVec.get());
        }
    }
}

void QEngineCUDA::UnlockSync()
{
    if (unlockHostMem) {
        clFinish();
        tryCuda("Failed to unmap buffer", [&] {
            return hipMemcpy(
                stateBuffer.get(), (void*)(stateVec.get()), sizeof(complex) * maxQPowerOcl, hipMemcpyHostToDevice);
        });
    } else {
        if (lockSyncFlags & CL_MAP_WRITE) {
            DISPATCH_BLOCK_WRITE(stateBuffer, 0U, sizeof(complex) * maxQPowerOcl, stateVec.get())
        }
        FreeStateVec();
    }

    lockSyncFlags = 0;
}

void QEngineCUDA::clFinish(bool doHard)
{
    if (!device_context) {
        return;
    }

    if (doHard) {
        hipDeviceSynchronize();
    } else {
        if (device_context->params_queue) {
            hipStreamSynchronize(device_context->params_queue);
        }
        if (device_context->queue) {
            hipStreamSynchronize(device_context->queue);
        }
    }

    wait_queue_items.clear();
}

void QEngineCUDA::clDump() { clFinish(); }

PoolItemPtr QEngineCUDA::GetFreePoolItem()
{
    std::lock_guard<std::mutex> lock(queue_mutex);

    while (wait_queue_items.size() >= poolItems.size()) {
        poolItems.push_back(std::make_shared<PoolItem>());
    }

    return poolItems[wait_queue_items.size()];
}

void QEngineCUDA::WaitCall(
    OCLAPI api_call, size_t workItemCount, size_t localGroupSize, std::vector<BufferPtr> args, size_t localBuffSize)
{
    QueueCall(api_call, workItemCount, localGroupSize, args, localBuffSize);
    clFinish();
}

void CUDART_CB _PopQueue(void* user_data) { ((QEngineCUDA*)user_data)->PopQueue(); }

void QEngineCUDA::PopQueue()
{
    std::lock_guard<std::mutex> lock(queue_mutex);

    if (poolItems.size()) {
        poolItems.front()->probArray = NULL;
        poolItems.front()->angleArray = NULL;

        if (poolItems.size() > 1) {
            rotate(poolItems.begin(), poolItems.begin() + 1, poolItems.end());
        }
    }

    if (!wait_queue_items.size()) {
        return;
    }

    QueueItem item = wait_queue_items.front();
    SubtractAlloc(item.deallocSize);
    if (item.isSetDoNorm) {
        doNormalize = item.doNorm;
    }
    if (item.isSetRunningNorm) {
        runningNorm = item.runningNorm;
    }

    wait_queue_items.pop_front();
}

void QEngineCUDA::DispatchQueue()
{
    QueueItem item;

    if (true) {
        std::lock_guard<std::mutex> lock(queue_mutex);

        if (!wait_queue_items.size()) {
            return;
        }

        item = wait_queue_items.back();

        if (item.isSetDoNorm || item.isSetRunningNorm) {
            hipLaunchHostFunc(device_context->queue, _PopQueue, (void*)this);
            return;
        }
    }

    std::vector<BufferPtr> args = item.buffers;

    // Dispatch the primary kernel, to apply the gate.
    switch (item.api_call) {
    case OCL_API_APPLY2X2:
        CUDA_KERNEL_4(apply2x2, qCudaCmplx, qCudaReal1, bitCapIntOcl, bitCapIntOcl);
        break;
    case OCL_API_APPLY2X2_SINGLE:
        CUDA_KERNEL_3(apply2x2single, qCudaCmplx, qCudaReal1, bitCapIntOcl);
        break;
    case OCL_API_APPLY2X2_NORM_SINGLE:
        CUDA_KERNEL_4(apply2x2normsingle, qCudaCmplx, qCudaReal1, bitCapIntOcl, qCudaReal1);
        break;
    case OCL_API_APPLY2X2_DOUBLE:
        CUDA_KERNEL_3(apply2x2double, qCudaCmplx, qCudaReal1, bitCapIntOcl);
        break;
    case OCL_API_APPLY2X2_WIDE:
        CUDA_KERNEL_4(apply2x2wide, qCudaCmplx, qCudaReal1, bitCapIntOcl, bitCapIntOcl);
        break;
    case OCL_API_APPLY2X2_SINGLE_WIDE:
        CUDA_KERNEL_3(apply2x2singlewide, qCudaCmplx, qCudaReal1, bitCapIntOcl);
        break;
    case OCL_API_APPLY2X2_NORM_SINGLE_WIDE:
        CUDA_KERNEL_4(apply2x2normsinglewide, qCudaCmplx, qCudaReal1, bitCapIntOcl, qCudaReal1);
        break;
    case OCL_API_APPLY2X2_DOUBLE_WIDE:
        CUDA_KERNEL_3(apply2x2doublewide, qCudaCmplx, qCudaReal1, bitCapIntOcl);
        break;
    case OCL_API_PHASE_SINGLE:
        CUDA_KERNEL_3(phasesingle, qCudaCmplx, qCudaCmplx, bitCapIntOcl);
        break;
    case OCL_API_PHASE_SINGLE_WIDE:
        CUDA_KERNEL_3(phasesinglewide, qCudaCmplx, qCudaCmplx, bitCapIntOcl);
        break;
    case OCL_API_INVERT_SINGLE:
        CUDA_KERNEL_3(invertsingle, qCudaCmplx, qCudaCmplx, bitCapIntOcl);
        break;
    case OCL_API_INVERT_SINGLE_WIDE:
        CUDA_KERNEL_3(invertsinglewide, qCudaCmplx, qCudaCmplx, bitCapIntOcl);
        break;
    case OCL_API_UNIFORMLYCONTROLLED:
        CUDA_KERNEL_5(uniformlycontrolled, qCudaCmplx, bitCapIntOcl, bitCapIntOcl, qCudaReal1, qCudaReal1);
        break;
    case OCL_API_UNIFORMPARITYRZ:
        CUDA_KERNEL_3(uniformparityrz, qCudaCmplx, bitCapIntOcl, qCudaCmplx);
        break;
    case OCL_API_UNIFORMPARITYRZ_NORM:
        CUDA_KERNEL_3(uniformparityrznorm, qCudaCmplx, bitCapIntOcl, qCudaCmplx);
        break;
    case OCL_API_CUNIFORMPARITYRZ:
        CUDA_KERNEL_4(cuniformparityrz, qCudaCmplx, bitCapIntOcl, qCudaCmplx, bitCapIntOcl);
        break;
    case OCL_API_COMPOSE:
        CUDA_KERNEL_4(compose, qCudaCmplx, qCudaCmplx, bitCapIntOcl, qCudaCmplx);
        break;
    case OCL_API_COMPOSE_WIDE:
        CUDA_KERNEL_4(composewide, qCudaCmplx, qCudaCmplx, bitCapIntOcl, qCudaCmplx);
        break;
    case OCL_API_COMPOSE_MID:
        CUDA_KERNEL_4(composemid, qCudaCmplx, qCudaCmplx, bitCapIntOcl, qCudaCmplx);
        break;
    case OCL_API_DECOMPOSEPROB:
        CUDA_KERNEL_6(decomposeprob, qCudaCmplx, bitCapIntOcl, qCudaReal1, qCudaReal1, qCudaReal1, qCudaReal1);
        break;
    case OCL_API_DECOMPOSEAMP:
        CUDA_KERNEL_4(decomposeamp, qCudaReal1, qCudaReal1, bitCapIntOcl, qCudaCmplx);
        break;
    case OCL_API_DISPOSEPROB:
        CUDA_KERNEL_4(disposeprob, qCudaCmplx, bitCapIntOcl, qCudaReal1, qCudaReal1);
        break;
    case OCL_API_DISPOSE:
        CUDA_KERNEL_3(dispose, qCudaCmplx, bitCapIntOcl, qCudaCmplx);
        break;
    case OCL_API_PROB:
        CUDA_KERNEL_3(prob, qCudaCmplx, bitCapIntOcl, qCudaReal1);
        break;
    case OCL_API_CPROB:
        CUDA_KERNEL_3(cprob, qCudaCmplx, bitCapIntOcl, qCudaReal1);
        break;
    case OCL_API_PROBREG:
        CUDA_KERNEL_3(probreg, qCudaCmplx, bitCapIntOcl, qCudaReal1);
        break;
    case OCL_API_PROBREGALL:
        CUDA_KERNEL_3(probregall, qCudaCmplx, bitCapIntOcl, qCudaReal1);
        break;
    case OCL_API_PROBMASK:
        CUDA_KERNEL_4(probmask, qCudaCmplx, bitCapIntOcl, qCudaReal1, bitCapIntOcl);
        break;
    case OCL_API_PROBMASKALL:
        CUDA_KERNEL_5(probmaskall, qCudaCmplx, bitCapIntOcl, qCudaReal1, bitCapIntOcl, bitCapIntOcl);
        break;
    case OCL_API_PROBPARITY:
        CUDA_KERNEL_3(probparity, qCudaCmplx, bitCapIntOcl, qCudaReal1);
        break;
    case OCL_API_FORCEMPARITY:
        CUDA_KERNEL_3(forcemparity, qCudaCmplx, bitCapIntOcl, qCudaReal1);
        break;
    case OCL_API_EXPPERM:
        CUDA_KERNEL_4(expperm, qCudaCmplx, bitCapIntOcl, bitCapIntOcl, qCudaReal1);
        break;
    case OCL_API_X_SINGLE:
        CUDA_KERNEL_2(xsingle, qCudaCmplx, bitCapIntOcl);
        break;
    case OCL_API_X_SINGLE_WIDE:
        CUDA_KERNEL_2(xsinglewide, qCudaCmplx, bitCapIntOcl);
        break;
    case OCL_API_X_MASK:
        CUDA_KERNEL_2(xmask, qCudaCmplx, bitCapIntOcl);
        break;
    case OCL_API_Z_SINGLE:
        CUDA_KERNEL_2(zsingle, qCudaCmplx, bitCapIntOcl);
        break;
    case OCL_API_Z_SINGLE_WIDE:
        CUDA_KERNEL_2(zsinglewide, qCudaCmplx, bitCapIntOcl);
        break;
    case OCL_API_PHASE_PARITY:
        CUDA_KERNEL_3(phaseparity, qCudaCmplx, bitCapIntOcl, qCudaCmplx);
        break;
    case OCL_API_ROL:
        CUDA_KERNEL_3(rol, qCudaCmplx, bitCapIntOcl, qCudaCmplx);
        break;
    case OCL_API_APPROXCOMPARE:
        CUDA_KERNEL_4(approxcompare, qCudaCmplx, qCudaCmplx, bitCapIntOcl, qCudaCmplx);
        break;
    case OCL_API_NORMALIZE:
        CUDA_KERNEL_3(nrmlze, qCudaCmplx, bitCapIntOcl, qCudaCmplx);
        break;
    case OCL_API_NORMALIZE_WIDE:
        CUDA_KERNEL_3(nrmlzewide, qCudaCmplx, bitCapIntOcl, qCudaCmplx);
        break;
    case OCL_API_UPDATENORM:
        CUDA_KERNEL_4(updatenorm, qCudaCmplx, bitCapIntOcl, qCudaReal1, qCudaReal1);
        break;
    case OCL_API_APPLYM:
        CUDA_KERNEL_3(applym, qCudaCmplx, bitCapIntOcl, qCudaCmplx);
        break;
    case OCL_API_APPLYMREG:
        CUDA_KERNEL_3(applymreg, qCudaCmplx, bitCapIntOcl, qCudaCmplx);
        break;
    case OCL_API_SHUFFLEBUFFERS:
        CUDA_KERNEL_3(shufflebuffers, qCudaCmplx, qCudaCmplx, bitCapIntOcl);
        break;
#if ENABLE_ALU
    case OCL_API_INC:
        CUDA_KERNEL_3(inc, qCudaCmplx, bitCapIntOcl, qCudaCmplx);
        break;
    case OCL_API_CINC:
        CUDA_KERNEL_4(cinc, qCudaCmplx, bitCapIntOcl, qCudaCmplx, bitCapIntOcl);
        break;
    case OCL_API_INCDECC:
        CUDA_KERNEL_3(incdecc, qCudaCmplx, bitCapIntOcl, qCudaCmplx);
        break;
    case OCL_API_INCS:
        CUDA_KERNEL_3(incs, qCudaCmplx, bitCapIntOcl, qCudaCmplx);
        break;
    case OCL_API_INCDECSC_1:
        CUDA_KERNEL_3(incdecsc1, qCudaCmplx, bitCapIntOcl, qCudaCmplx);
        break;
    case OCL_API_INCDECSC_2:
        CUDA_KERNEL_3(incdecsc2, qCudaCmplx, bitCapIntOcl, qCudaCmplx);
        break;
    case OCL_API_MUL:
        CUDA_KERNEL_3(mul, qCudaCmplx, bitCapIntOcl, qCudaCmplx);
        break;
    case OCL_API_DIV:
        CUDA_KERNEL_3(div, qCudaCmplx, bitCapIntOcl, qCudaCmplx);
        break;
    case OCL_API_MULMODN_OUT:
        CUDA_KERNEL_3(mulmodnout, qCudaCmplx, bitCapIntOcl, qCudaCmplx);
        break;
    case OCL_API_IMULMODN_OUT:
        CUDA_KERNEL_3(imulmodnout, qCudaCmplx, bitCapIntOcl, qCudaCmplx);
        break;
    case OCL_API_POWMODN_OUT:
        CUDA_KERNEL_3(powmodnout, qCudaCmplx, bitCapIntOcl, qCudaCmplx);
        break;
    case OCL_API_CMUL:
        CUDA_KERNEL_4(cmul, qCudaCmplx, bitCapIntOcl, qCudaCmplx, bitCapIntOcl);
        break;
    case OCL_API_CDIV:
        CUDA_KERNEL_4(cdiv, qCudaCmplx, bitCapIntOcl, qCudaCmplx, bitCapIntOcl);
        break;
    case OCL_API_CMULMODN_OUT:
        CUDA_KERNEL_4(cmulmodnout, qCudaCmplx, bitCapIntOcl, qCudaCmplx, bitCapIntOcl);
        break;
    case OCL_API_CIMULMODN_OUT:
        CUDA_KERNEL_4(cimulmodnout, qCudaCmplx, bitCapIntOcl, qCudaCmplx, bitCapIntOcl);
        break;
    case OCL_API_CPOWMODN_OUT:
        CUDA_KERNEL_4(cpowmodnout, qCudaCmplx, bitCapIntOcl, qCudaCmplx, bitCapIntOcl);
        break;
    case OCL_API_FULLADD:
        CUDA_KERNEL_2(fulladd, qCudaCmplx, bitCapIntOcl);
        break;
    case OCL_API_IFULLADD:
        CUDA_KERNEL_2(ifulladd, qCudaCmplx, bitCapIntOcl);
        break;
    case OCL_API_INDEXEDLDA:
        CUDA_KERNEL_4(indexedLda, qCudaCmplx, bitCapIntOcl, qCudaCmplx, unsigned char);
        break;
    case OCL_API_INDEXEDADC:
        CUDA_KERNEL_4(indexedAdc, qCudaCmplx, bitCapIntOcl, qCudaCmplx, unsigned char);
        break;
    case OCL_API_INDEXEDSBC:
        CUDA_KERNEL_4(indexedSbc, qCudaCmplx, bitCapIntOcl, qCudaCmplx, unsigned char);
        break;
    case OCL_API_HASH:
        CUDA_KERNEL_4(hash, qCudaCmplx, bitCapIntOcl, qCudaCmplx, unsigned char);
        break;
    case OCL_API_CPHASEFLIPIFLESS:
        CUDA_KERNEL_2(cphaseflipifless, qCudaCmplx, bitCapIntOcl);
        break;
    case OCL_API_PHASEFLIPIFLESS:
        CUDA_KERNEL_2(phaseflipifless, qCudaCmplx, bitCapIntOcl);
        break;
#if ENABLE_BCD
    case OCL_API_INCBCD:
        CUDA_KERNEL_3(incbcd, qCudaCmplx, bitCapIntOcl, qCudaCmplx);
        break;
    case OCL_API_INCDECBCDC:
        CUDA_KERNEL_3(incdecbcdc, qCudaCmplx, bitCapIntOcl, qCudaCmplx);
        break;
#endif
#endif
    case OCL_API_UNKNOWN:
    default:
        throw std::runtime_error("Invalid CUDA kernel selected!");
    }

    hipLaunchHostFunc(device_context->queue, _PopQueue, (void*)this);
}

void QEngineCUDA::SetDevice(int64_t dID)
{
    const size_t deviceCount = CUDAEngine::Instance().GetDeviceCount();

    if (!deviceCount) {
        throw std::runtime_error("QEngineCUDA::SetDevice(): No available devices.");
    }

    if (dID > ((int64_t)deviceCount)) {
        throw std::runtime_error("QEngineCUDA::SetDevice(): Requested device doesn't exist.");
    }

    clFinish();

    const DeviceContextPtr nDeviceContext = CUDAEngine::Instance().GetDeviceContextPtr(dID);
    const int64_t defDevId = (int)CUDAEngine::Instance().GetDefaultDeviceID();

    if (!didInit) {
        AddAlloc(sizeof(complex) * maxQPowerOcl);
    } else if ((dID == deviceID) || ((dID == -1) && (deviceID == defDevId)) ||
        ((deviceID == -1) && (dID == defDevId))) {
        // If we're "switching" to the device we already have, don't reinitialize.
        return;
    }

    device_context = nDeviceContext;
    deviceID = dID;

    // If the user wants not to use host RAM, but we can't allocate enough on the device, fall back to host RAM anyway.
    const size_t stateVecSize = maxQPowerOcl * sizeof(complex);
#if ENABLE_OCL_MEM_GUARDS
    // Device RAM should be large enough for 2 times the size of the stateVec, plus some excess.
    if (stateVecSize > device_context->GetMaxAlloc()) {
        throw bad_alloc("VRAM limits exceeded in QEngineCUDA::SetDevice()");
    }
#endif
    usingHostRam = (useHostRam || ((OclMemDenom * stateVecSize) > device_context->GetGlobalSize()));

    const bitCapIntOcl oldNrmVecAlignSize = nrmGroupSize ? (nrmGroupCount / nrmGroupSize) : 0U;
    nrmGroupCount = device_context->GetPreferredConcurrency();
    nrmGroupSize = device_context->GetPreferredSizeMultiple();
    if (nrmGroupSize > device_context->GetMaxWorkGroupSize()) {
        nrmGroupSize = device_context->GetMaxWorkGroupSize();
    }
    // constrain to a power of two
    nrmGroupSize = pow2Ocl(log2Ocl(nrmGroupSize));

    const size_t nrmArrayAllocSize =
        (!nrmGroupSize || ((sizeof(real1) * nrmGroupCount / nrmGroupSize) < QRACK_ALIGN_SIZE))
        ? QRACK_ALIGN_SIZE
        : (sizeof(real1) * nrmGroupCount / nrmGroupSize);

    const bool doResize = (nrmGroupCount / nrmGroupSize) != oldNrmVecAlignSize;

    nrmBuffer = NULL;
    if (didInit && doResize) {
        nrmArray = NULL;
        SubtractAlloc(oldNrmVecAlignSize);
    }

    if (!didInit || doResize) {
        AddAlloc(nrmArrayAllocSize);
#if defined(__ANDROID__)
        nrmArray = std::unique_ptr<real1[], void (*)(real1*)>(
            new real1[nrmArrayAllocSize / sizeof(real1)], [](real1* r) { delete[] r; });
#elif defined(__APPLE__)
        nrmArray = std::unique_ptr<real1[], void (*)(real1*)>(
            _aligned_nrm_array_alloc(nrmArrayAllocSize), [](real1* c) { free(c); });
#elif defined(_WIN32) && !defined(__CYGWIN__)
        nrmArray = std::unique_ptr<real1[], void (*)(real1*)>(
            (real1*)_aligned_malloc(nrmArrayAllocSize, QRACK_ALIGN_SIZE), [](real1* c) { _aligned_free(c); });
#else
        nrmArray = std::unique_ptr<real1[], void (*)(real1*)>(
            (real1*)aligned_alloc(QRACK_ALIGN_SIZE, nrmArrayAllocSize), [](real1* c) { free(c); });
#endif
    }
    nrmBuffer = MakeBuffer(CL_MEM_READ_WRITE, nrmArrayAllocSize);

    poolItems.clear();
    poolItems.push_back(std::make_shared<PoolItem>());

    if (!didInit) {
        stateVec = AllocStateVec(maxQPowerOcl, usingHostRam);
        stateBuffer = MakeStateVecBuffer(stateVec);
    }

    didInit = true;
}

real1_f QEngineCUDA::ParSum(real1* toSum, bitCapIntOcl maxI)
{
    // This interface is potentially parallelizable, but, for now, better performance is probably given by implementing
    // it as a serial loop.
    real1 totSum = ZERO_R1;
    for (bitCapIntOcl i = 0U; i < maxI; ++i) {
        totSum += toSum[i];
    }

    return (real1_f)totSum;
}

void QEngineCUDA::InitOCL(int64_t devID) { SetDevice(devID); }

void QEngineCUDA::ResetStateBuffer(BufferPtr nStateBuffer) { stateBuffer = nStateBuffer; }

void QEngineCUDA::SetPermutation(bitCapInt perm, complex phaseFac)
{
    clDump();

    if (!stateBuffer) {
        ReinitBuffer();
    }

    ClearBuffer(stateBuffer, 0U, maxQPowerOcl);

    // If "permutationAmp" amp is in (read-only) use, this method completely supersedes that application anyway.

    if (phaseFac == CMPLX_DEFAULT_ARG) {
        permutationAmp = GetNonunitaryPhase();
    } else {
        permutationAmp = phaseFac;
    }

    tryCuda("Failed to enqueue buffer write", [&] {
        return hipMemcpy((void*)((complex*)(stateBuffer.get()) + perm.bits[0U]), (void*)&permutationAmp,
            sizeof(complex), hipMemcpyHostToDevice);
    });

    QueueSetRunningNorm(ONE_R1_F);
}

/// NOT gate, which is also Pauli x matrix
void QEngineCUDA::X(bitLenInt qubit)
{
    const complex pauliX[4]{ ZERO_CMPLX, ONE_CMPLX, ONE_CMPLX, ZERO_CMPLX };
    const bitCapIntOcl qPowers[1]{ pow2Ocl(qubit) };
    Apply2x2(0U, qPowers[0], pauliX, 1U, qPowers, false, SPECIAL_2X2::PAULIX);
}

/// Apply Pauli Z matrix to bit
void QEngineCUDA::Z(bitLenInt qubit)
{
    const complex pauliZ[4]{ ONE_CMPLX, ZERO_CMPLX, ZERO_CMPLX, -ONE_CMPLX };
    const bitCapIntOcl qPowers[1]{ pow2Ocl(qubit) };
    Apply2x2(0U, qPowers[0], pauliZ, 1U, qPowers, false, SPECIAL_2X2::PAULIZ);
}

void QEngineCUDA::Invert(complex topRight, complex bottomLeft, bitLenInt qubitIndex)
{
    if ((randGlobalPhase || IS_NORM_0(ONE_CMPLX - topRight)) && IS_NORM_0(topRight - bottomLeft)) {
        X(qubitIndex);
        return;
    }

    const complex pauliX[4]{ ZERO_CMPLX, topRight, bottomLeft, ZERO_CMPLX };
    const bitCapIntOcl qPowers[1]{ pow2Ocl(qubitIndex) };
    Apply2x2(0U, qPowers[0], pauliX, 1U, qPowers, false, SPECIAL_2X2::INVERT);
}

void QEngineCUDA::Phase(complex topLeft, complex bottomRight, bitLenInt qubitIndex)
{
    if (randGlobalPhase || IS_NORM_0(ONE_CMPLX - topLeft)) {
        if (IS_NORM_0(topLeft - bottomRight)) {
            return;
        }

        if (IS_NORM_0(topLeft + bottomRight)) {
            Z(qubitIndex);
            return;
        }
    }

    const complex pauliZ[4]{ topLeft, ZERO_CMPLX, ZERO_CMPLX, bottomRight };
    const bitCapIntOcl qPowers[1]{ pow2Ocl(qubitIndex) };
    Apply2x2(0U, qPowers[0], pauliZ, 1U, qPowers, false, SPECIAL_2X2::PHASE);
}

void QEngineCUDA::XMask(bitCapInt mask)
{
    if (bi_compare_0(mask) == 0) {
        return;
    }
    if (isPowerOfTwo(mask)) {
        X(log2(mask));
        return;
    }

    BitMask(mask.bits[0U], OCL_API_X_MASK);
}

void QEngineCUDA::PhaseParity(real1_f radians, bitCapInt mask)
{
    if (bi_compare_0(mask) == 0) {
        return;
    }

    if (isPowerOfTwo(mask)) {
        complex phaseFac = std::polar(ONE_R1, (real1)(radians / 2));
        Phase(ONE_CMPLX / phaseFac, phaseFac, log2(mask));
        return;
    }

    BitMask(mask.bits[0U], OCL_API_PHASE_PARITY, radians);
}

void QEngineCUDA::Apply2x2(bitCapIntOcl offset1, bitCapIntOcl offset2, const complex* mtrx, bitLenInt bitCount,
    const bitCapIntOcl* qPowersSorted, bool doCalcNorm, SPECIAL_2X2 special, real1_f norm_thresh)
{
    CHECK_ZERO_SKIP();

    if ((offset1 >= maxQPowerOcl) || (offset2 >= maxQPowerOcl)) {
        throw std::invalid_argument(
            "QEngineCUDA::Apply2x2 offset1 and offset2 parameters must be within allocated qubit bounds!");
    }

    for (bitLenInt i = 0U; i < bitCount; ++i) {
        if (qPowersSorted[i] >= maxQPowerOcl) {
            throw std::invalid_argument(
                "QEngineCUDA::Apply2x2 parameter qPowersSorted array values must be within allocated qubit bounds!");
        }
    }

    const bool skipNorm = !doNormalize || (abs(ONE_R1 - runningNorm) <= FP_NORM_EPSILON);
    const bool isXGate = skipNorm && (special == SPECIAL_2X2::PAULIX);
    const bool isZGate = skipNorm && (special == SPECIAL_2X2::PAULIZ);
    const bool isInvertGate = skipNorm && (special == SPECIAL_2X2::INVERT);
    const bool isPhaseGate = skipNorm && (special == SPECIAL_2X2::PHASE);

    // Are we going to calculate the normalization factor, on the fly? We can't, if this call doesn't iterate through
    // every single permutation amplitude.
    bool doApplyNorm = doNormalize && (bitCount == 1) && (runningNorm > ZERO_R1) && !isXGate && !isZGate &&
        !isInvertGate && !isPhaseGate;
    doCalcNorm &= doApplyNorm || (runningNorm <= ZERO_R1);
    doApplyNorm &= (runningNorm != ONE_R1);

    PoolItemPtr poolItem = GetFreePoolItem();

    // Arguments are concatenated into buffers by primitive type, such as integer or complex number.

    // Load the integer kernel arguments buffer.
    const bitCapIntOcl maxI = maxQPowerOcl >> bitCount;
    bitCapIntOcl bciArgs[5]{ offset2, offset1, maxI, bitCount, 0U };

    // We have default OpenCL work item counts and group sizes, but we may need to use different values due to the total
    // amount of work in this method call instance.
    const size_t ngc = FixWorkItemCount(maxI, nrmGroupCount);
    const size_t ngs = FixGroupSize(ngc, nrmGroupSize);

    // In an efficient OpenCL kernel, every single byte loaded comes at a significant execution time premium.
    // We handle single and double bit gates as special cases, for many reasons. Given that we have already separated
    // these out as special cases, since we know the bit count, we can eliminate the qPowersSorted buffer, by loading
    // its one or two values into the bciArgs buffer, of the same type. This gives us a significant execution time
    // savings.
    size_t bciArgsSize = 4;
    if (bitCount == 1) {
        // Single bit gates offsets are always 0 and target bit power. Hence, we overwrite one of the bit offset
        // arguments.
        if (ngc == maxI) {
            bciArgsSize = 3;
            bciArgs[2] = qPowersSorted[0] - 1U;
        } else {
            bciArgsSize = 4;
            bciArgs[3] = qPowersSorted[0] - 1U;
        }
    } else if (bitCount == 2) {
        // Double bit gates include both controlled and swap gates. To reuse the code for both cases, we need two offset
        // arguments. Hence, we cannot easily overwrite either of the bit offset arguments.
        bciArgsSize = 5;
        bciArgs[3] = qPowersSorted[0] - 1U;
        bciArgs[4] = qPowersSorted[1] - 1U;
    }
    DISPATCH_TEMP_WRITE(poolItem->ulongBuffer, sizeof(bitCapIntOcl) * bciArgsSize, bciArgs);

    // Load the 2x2 complex matrix and the normalization factor into the complex arguments buffer.
    complex cmplx[CMPLX_NORM_LEN];
    std::copy(mtrx, mtrx + 4, cmplx);

    // Is the vector already normalized, or is this method not appropriate for on-the-fly normalization?
    cmplx[4] = complex(doApplyNorm ? (ONE_R1 / (real1)sqrt(runningNorm)) : ONE_R1, ZERO_R1);
    cmplx[5] = (real1)norm_thresh;

    BufferPtr locCmplxBuffer;
    if (!isXGate && !isZGate) {
        DISPATCH_TEMP_WRITE(poolItem->cmplxBuffer, sizeof(complex) * CMPLX_NORM_LEN, cmplx);
    }

    // Load a buffer with the powers of 2 of each bit index involved in the operation.
    BufferPtr locPowersBuffer;
    if (bitCount > 2) {
        locPowersBuffer = MakeBuffer(CL_MEM_READ_ONLY, sizeof(bitCapIntOcl) * bitCount);
        if (sizeof(bitCapInt) == sizeof(bitCapIntOcl)) {
            DISPATCH_TEMP_WRITE(locPowersBuffer, sizeof(bitCapIntOcl) * bitCount, qPowersSorted);
        } else {
            DISPATCH_TEMP_WRITE(locPowersBuffer, sizeof(bitCapIntOcl) * bitCount, qPowersSorted);
        }
    }

    // We load the appropriate kernel, that does/doesn't CALCULATE the norm, and does/doesn't APPLY the norm.
    unsigned char kernelMask = APPLY2X2_DEFAULT;
    if (bitCount == 1) {
        kernelMask |= APPLY2X2_SINGLE;
        if (isXGate) {
            kernelMask |= APPLY2X2_X;
        } else if (isZGate) {
            kernelMask |= APPLY2X2_Z;
        } else if (isInvertGate) {
            kernelMask |= APPLY2X2_INVERT;
        } else if (isPhaseGate) {
            kernelMask |= APPLY2X2_PHASE;
        } else if (doCalcNorm) {
            kernelMask |= APPLY2X2_NORM;
        }
    } else if (bitCount == 2) {
        kernelMask |= APPLY2X2_DOUBLE;
    }
    if (ngc == maxI) {
        kernelMask |= APPLY2X2_WIDE;
    }

    OCLAPI api_call;
    switch (kernelMask) {
    case APPLY2X2_DEFAULT:
        api_call = OCL_API_APPLY2X2;
        break;
    case APPLY2X2_SINGLE:
        api_call = OCL_API_APPLY2X2_SINGLE;
        break;
    case APPLY2X2_SINGLE | APPLY2X2_X:
        api_call = OCL_API_X_SINGLE;
        break;
    case APPLY2X2_SINGLE | APPLY2X2_Z:
        api_call = OCL_API_Z_SINGLE;
        break;
    case APPLY2X2_SINGLE | APPLY2X2_INVERT:
        api_call = OCL_API_INVERT_SINGLE;
        break;
    case APPLY2X2_SINGLE | APPLY2X2_PHASE:
        api_call = OCL_API_PHASE_SINGLE;
        break;
    case APPLY2X2_NORM | APPLY2X2_SINGLE:
        api_call = OCL_API_APPLY2X2_NORM_SINGLE;
        break;
    case APPLY2X2_DOUBLE:
        api_call = OCL_API_APPLY2X2_DOUBLE;
        break;
    case APPLY2X2_WIDE:
        api_call = OCL_API_APPLY2X2_WIDE;
        break;
    case APPLY2X2_SINGLE | APPLY2X2_WIDE:
        api_call = OCL_API_APPLY2X2_SINGLE_WIDE;
        break;
    case APPLY2X2_SINGLE | APPLY2X2_WIDE | APPLY2X2_X:
        api_call = OCL_API_X_SINGLE_WIDE;
        break;
    case APPLY2X2_SINGLE | APPLY2X2_WIDE | APPLY2X2_Z:
        api_call = OCL_API_Z_SINGLE_WIDE;
        break;
    case APPLY2X2_SINGLE | APPLY2X2_WIDE | APPLY2X2_INVERT:
        api_call = OCL_API_INVERT_SINGLE_WIDE;
        break;
    case APPLY2X2_SINGLE | APPLY2X2_WIDE | APPLY2X2_PHASE:
        api_call = OCL_API_PHASE_SINGLE_WIDE;
        break;
    case APPLY2X2_NORM | APPLY2X2_SINGLE | APPLY2X2_WIDE:
        api_call = OCL_API_APPLY2X2_NORM_SINGLE_WIDE;
        break;
    case APPLY2X2_DOUBLE | APPLY2X2_WIDE:
        api_call = OCL_API_APPLY2X2_DOUBLE_WIDE;
        break;
    default:
        throw std::runtime_error("Invalid APPLY2X2 kernel selected!");
    }

    if (isXGate || isZGate) {
        QueueCall(api_call, ngc, ngs, { stateBuffer, poolItem->ulongBuffer });
    } else if (doCalcNorm) {
        if (bitCount > 2) {
            QueueCall(api_call, ngc, ngs,
                { stateBuffer, poolItem->cmplxBuffer, poolItem->ulongBuffer, locPowersBuffer, nrmBuffer },
                sizeof(real1) * ngs);
        } else {
            QueueCall(api_call, ngc, ngs, { stateBuffer, poolItem->cmplxBuffer, poolItem->ulongBuffer, nrmBuffer },
                sizeof(real1) * ngs);
        }
    } else {
        if (bitCount > 2) {
            QueueCall(
                api_call, ngc, ngs, { stateBuffer, poolItem->cmplxBuffer, poolItem->ulongBuffer, locPowersBuffer });
        } else {
            QueueCall(api_call, ngc, ngs, { stateBuffer, poolItem->cmplxBuffer, poolItem->ulongBuffer });
        }
    }

    if (doApplyNorm) {
        QueueSetRunningNorm(ONE_R1_F);
    }

    if (!doCalcNorm) {
        return;
    }

    // If we have calculated the norm of the state vector in this call, we need to sum the buffer of partial norm
    // values into a single normalization constant.
    WAIT_REAL1_SUM(nrmBuffer, ngc / ngs, nrmArray, &runningNorm);
    if (runningNorm <= FP_NORM_EPSILON) {
        ZeroAmplitudes();
    }
}

void QEngineCUDA::BitMask(bitCapIntOcl mask, OCLAPI api_call, real1_f phase)
{
    if (mask >= maxQPowerOcl) {
        throw std::invalid_argument("QEngineCUDA::BitMask mask out-of-bounds!");
    }

    CHECK_ZERO_SKIP();

    bitCapIntOcl otherMask = (maxQPowerOcl - 1U) ^ mask;

    PoolItemPtr poolItem = GetFreePoolItem();

    const bitCapIntOcl bciArgs[BCI_ARG_LEN]{ maxQPowerOcl, mask, otherMask, 0U, 0U, 0U, 0U, 0U, 0U, 0U };

    DISPATCH_TEMP_WRITE(poolItem->ulongBuffer, sizeof(bitCapIntOcl) * 3, bciArgs);

    const size_t ngc = FixWorkItemCount(bciArgs[0], nrmGroupCount);
    const size_t ngs = FixGroupSize(ngc, nrmGroupSize);

    const bool isPhaseParity = (api_call == OCL_API_PHASE_PARITY);
    if (isPhaseParity) {
        const complex phaseFac = std::polar(ONE_R1, (real1)(phase / 2));
        const complex cmplxArray[2]{ phaseFac, ONE_CMPLX / phaseFac };
        DISPATCH_TEMP_WRITE(poolItem->cmplxBuffer, 2U * sizeof(complex), cmplxArray);
    }

    if (isPhaseParity) {
        QueueCall(api_call, ngc, ngs, { stateBuffer, poolItem->ulongBuffer, poolItem->cmplxBuffer });
    } else {
        QueueCall(api_call, ngc, ngs, { stateBuffer, poolItem->ulongBuffer });
    }
}

void QEngineCUDA::UniformlyControlledSingleBit(const std::vector<bitLenInt>& controls, bitLenInt qubitIndex,
    const complex* mtrxs, const std::vector<bitCapInt>& mtrxSkipPowers, bitCapInt mtrxSkipValueMask)
{
    CHECK_ZERO_SKIP();

    // If there are no controls, the base case should be the non-controlled single bit gate.
    if (!controls.size()) {
        Mtrx(mtrxs + (mtrxSkipValueMask.bits[0U] << 2U), qubitIndex);
        return;
    }

    if (qubitIndex >= qubitCount) {
        throw std::invalid_argument("QEngineCUDA::UniformlyControlledSingleBit qubitIndex is out-of-bounds!");
    }

    ThrowIfQbIdArrayIsBad(controls, qubitCount, "QEngineCUDA::UniformlyControlledSingleBit control is out-of-bounds!");

    // We grab the wait event queue. We will replace it with three new asynchronous events, to wait for.
    PoolItemPtr poolItem = GetFreePoolItem();

    // Arguments are concatenated into buffers by primitive type, such as integer or complex number.

    // Load the integer kernel arguments buffer.
    const bitCapIntOcl maxI = maxQPowerOcl >> 1U;
    const bitCapIntOcl bciArgs[BCI_ARG_LEN]{ maxI, pow2Ocl(qubitIndex), (bitCapIntOcl)controls.size(),
        (bitCapIntOcl)mtrxSkipPowers.size(), mtrxSkipValueMask.bits[0U], 0U, 0U, 0U, 0U, 0U };
    DISPATCH_WRITE(poolItem->ulongBuffer, sizeof(bitCapIntOcl) * 5, bciArgs);

    BufferPtr nrmInBuffer = MakeBuffer(CL_MEM_READ_ONLY, sizeof(real1));
    const real1 nrm = (runningNorm > ZERO_R1) ? ONE_R1 / (real1)sqrt(runningNorm) : ONE_R1;
    DISPATCH_WRITE(nrmInBuffer, sizeof(real1), &nrm);

    const size_t sizeDiff = sizeof(complex) * pow2Ocl(controls.size() + mtrxSkipPowers.size()) << 2U;
    AddAlloc(sizeDiff);
    BufferPtr uniformBuffer = MakeBuffer(CL_MEM_READ_ONLY, sizeDiff);

    DISPATCH_WRITE(uniformBuffer, sizeof(complex) * pow2Ocl(controls.size() + mtrxSkipPowers.size()) << 2U, mtrxs);

    std::unique_ptr<bitCapIntOcl[]> qPowers(new bitCapIntOcl[controls.size() + mtrxSkipPowers.size()]);
    std::transform(controls.begin(), controls.end(), qPowers.get(), pow2Ocl);
    std::transform(mtrxSkipPowers.begin(), mtrxSkipPowers.end(), qPowers.get() + controls.size(),
        [](bitCapInt i) { return i.bits[0U]; });

    // We have default OpenCL work item counts and group sizes, but we may need to use different values due to the total
    // amount of work in this method call instance.
    const size_t ngc = FixWorkItemCount(maxI, nrmGroupCount);
    const size_t ngs = FixGroupSize(ngc, nrmGroupSize);

    const size_t powBuffSize = sizeof(bitCapIntOcl) * (controls.size() + mtrxSkipPowers.size());
    AddAlloc(powBuffSize);
    BufferPtr powersBuffer = MakeBuffer(CL_MEM_READ_ONLY, sizeof(bitCapIntOcl) * pow2Ocl(QBCAPPOW));

    // Load a buffer with the powers of 2 of each bit index involved in the operation.
    DISPATCH_WRITE(powersBuffer, powBuffSize, qPowers.get());

    // We call the kernel, with global buffers and one local buffer.
    WaitCall(OCL_API_UNIFORMLYCONTROLLED, ngc, ngs,
        { stateBuffer, poolItem->ulongBuffer, powersBuffer, uniformBuffer, nrmInBuffer });

    uniformBuffer.reset();
    qPowers.reset();

    SubtractAlloc(sizeDiff + powBuffSize);

    runningNorm = ONE_R1;
}

void QEngineCUDA::UniformParityRZ(bitCapInt mask, real1_f angle)
{
    if (bi_compare(mask, maxQPower) >= 0) {
        throw std::invalid_argument("QEngineCUDA::UniformParityRZ mask out-of-bounds!");
    }

    CHECK_ZERO_SKIP();

    const bitCapIntOcl bciArgs[BCI_ARG_LEN]{ maxQPowerOcl, mask.bits[0U], 0U, 0U, 0U, 0U, 0U, 0U, 0U, 0U };
    const real1 cosine = (real1)cos(angle);
    const real1 sine = (real1)sin(angle);
    const complex phaseFacs[3]{ complex(cosine, sine), complex(cosine, -sine),
        (runningNorm > ZERO_R1) ? (ONE_R1 / (real1)sqrt(runningNorm)) : ONE_R1 };

    PoolItemPtr poolItem = GetFreePoolItem();

    DISPATCH_TEMP_WRITE(poolItem->ulongBuffer, sizeof(bitCapIntOcl) * 2, bciArgs);
    DISPATCH_TEMP_WRITE(poolItem->cmplxBuffer, sizeof(complex) * 3, &phaseFacs);

    const size_t ngc = FixWorkItemCount(bciArgs[0], nrmGroupCount);
    const size_t ngs = FixGroupSize(ngc, nrmGroupSize);

    QueueCall((abs(ONE_R1 - runningNorm) <= FP_NORM_EPSILON) ? OCL_API_UNIFORMPARITYRZ : OCL_API_UNIFORMPARITYRZ_NORM,
        ngc, ngs, { stateBuffer, poolItem->ulongBuffer, poolItem->cmplxBuffer });
    QueueSetRunningNorm(ONE_R1_F);
}

void QEngineCUDA::CUniformParityRZ(const std::vector<bitLenInt>& controls, bitCapInt mask, real1_f angle)
{
    if (!controls.size()) {
        UniformParityRZ(mask, angle);
        return;
    }

    if (bi_compare(mask, maxQPowerOcl) >= 0) {
        throw std::invalid_argument("QEngineCUDA::CUniformParityRZ mask out-of-bounds!");
    }

    ThrowIfQbIdArrayIsBad(controls, qubitCount, "QEngineCUDA::CUniformParityRZ control is out-of-bounds!");

    CHECK_ZERO_SKIP();

    bitCapIntOcl controlMask = 0U;
    std::unique_ptr<bitCapIntOcl[]> controlPowers(new bitCapIntOcl[controls.size()]);
    for (bitLenInt i = 0U; i < controls.size(); ++i) {
        controlPowers[i] = pow2Ocl(controls[i]);
        controlMask |= controlPowers[i];
    }
    std::sort(controlPowers.get(), controlPowers.get() + controls.size());
    BufferPtr controlBuffer = MakeBuffer(
        CL_MEM_COPY_HOST_PTR | CL_MEM_READ_ONLY, sizeof(bitCapIntOcl) * controls.size(), controlPowers.get());
    controlPowers.reset();

    const bitCapIntOcl bciArgs[BCI_ARG_LEN]{ maxQPowerOcl >> (bitLenInt)controls.size(), mask.bits[0U], controlMask,
        (bitCapIntOcl)controls.size(), 0U, 0U, 0U, 0U, 0U, 0U };
    const real1 cosine = (real1)cos(angle);
    const real1 sine = (real1)sin(angle);
    const complex phaseFacs[2]{ complex(cosine, sine), complex(cosine, -sine) };

    PoolItemPtr poolItem = GetFreePoolItem();

    DISPATCH_TEMP_WRITE(poolItem->ulongBuffer, sizeof(bitCapIntOcl) << 2U, bciArgs);
    DISPATCH_TEMP_WRITE(poolItem->cmplxBuffer, sizeof(complex) << 1U, &phaseFacs);

    const size_t ngc = FixWorkItemCount(bciArgs[0], nrmGroupCount);
    const size_t ngs = FixGroupSize(ngc, nrmGroupSize);
    QueueCall(OCL_API_CUNIFORMPARITYRZ, ngc, ngs,
        { stateBuffer, poolItem->ulongBuffer, poolItem->cmplxBuffer, controlBuffer });
    QueueSetRunningNorm(ONE_R1_F);
}

void QEngineCUDA::ApplyMx(OCLAPI api_call, const bitCapIntOcl* bciArgs, complex nrm)
{
    CHECK_ZERO_SKIP();

    PoolItemPtr poolItem = GetFreePoolItem();

    DISPATCH_TEMP_WRITE(poolItem->ulongBuffer, sizeof(bitCapIntOcl) * 3, bciArgs);
    BufferPtr locCmplxBuffer = MakeBuffer(CL_MEM_READ_ONLY, sizeof(complex));
    DISPATCH_TEMP_WRITE(poolItem->cmplxBuffer, sizeof(complex), &nrm);

    const size_t ngc = FixWorkItemCount(bciArgs[0], nrmGroupCount);
    const size_t ngs = FixGroupSize(ngc, nrmGroupSize);

    QueueCall(api_call, ngc, ngs, { stateBuffer, poolItem->ulongBuffer, poolItem->cmplxBuffer });
    QueueSetRunningNorm(ONE_R1_F);
}

void QEngineCUDA::ApplyM(bitCapInt qPower, bool result, complex nrm)
{
    bitCapIntOcl powerTest = result ? qPower.bits[0U] : 0U;

    const bitCapIntOcl bciArgs[BCI_ARG_LEN]{ maxQPowerOcl >> 1U, qPower.bits[0U], powerTest, 0U, 0U, 0U, 0U, 0U, 0U,
        0U };

    ApplyMx(OCL_API_APPLYM, bciArgs, nrm);
}

void QEngineCUDA::ApplyM(bitCapInt mask, bitCapInt result, complex nrm)
{
    if (bi_compare(mask, maxQPowerOcl) >= 0) {
        throw std::invalid_argument("QEngineCUDA::ApplyM mask out-of-bounds!");
    }

    const bitCapIntOcl bciArgs[BCI_ARG_LEN]{ maxQPowerOcl, mask.bits[0U], result.bits[0U], 0U, 0U, 0U, 0U, 0U, 0U, 0U };

    ApplyMx(OCL_API_APPLYMREG, bciArgs, nrm);
}

void QEngineCUDA::Compose(OCLAPI apiCall, const bitCapIntOcl* bciArgs, QEngineCUDAPtr toCopy)
{
    if (!toCopy->qubitCount) {
        return;
    }

    if (!stateBuffer || !toCopy->stateBuffer) {
        // Compose will have a wider but 0 stateVec
        ZeroAmplitudes();
        SetQubitCount(qubitCount + toCopy->qubitCount);
        return;
    }

    if (!qubitCount) {
        clFinish();
        SetQubitCount(toCopy->qubitCount);
        toCopy->clFinish();
        runningNorm = toCopy->runningNorm;
        stateVec = AllocStateVec(toCopy->maxQPowerOcl, usingHostRam);
        stateBuffer = MakeStateVecBuffer(stateVec);

        tryCuda("Failed to enqueue buffer copy", [&] {
            return hipMemcpy(
                stateBuffer.get(), toCopy->stateBuffer.get(), sizeof(complex) * maxQPowerOcl, hipMemcpyDeviceToDevice);
        });

        return;
    }

    const bitCapIntOcl oMaxQPower = maxQPowerOcl;
    const bitCapIntOcl nMaxQPower = bciArgs[0];
    const bitCapIntOcl nQubitCount = bciArgs[1] + toCopy->qubitCount;
    const size_t nStateVecSize = nMaxQPower * sizeof(complex);
#if ENABLE_OCL_MEM_GUARDS
    if (nStateVecSize > device_context->GetMaxAlloc()) {
        throw bad_alloc("VRAM limits exceeded in QEngineCUDA::Compose()");
    }
#endif

    if (doNormalize) {
        NormalizeState();
    }
    if (toCopy->doNormalize) {
        toCopy->NormalizeState();
    }

    PoolItemPtr poolItem = GetFreePoolItem();

    DISPATCH_TEMP_WRITE(poolItem->ulongBuffer, sizeof(bitCapIntOcl) * 7, bciArgs);

    AddAlloc(sizeof(complex) * nMaxQPower);

    SetQubitCount(nQubitCount);

    const size_t ngc = FixWorkItemCount(maxQPowerOcl, nrmGroupCount);
    const size_t ngs = FixGroupSize(ngc, nrmGroupSize);

    std::shared_ptr<complex> nStateVec = AllocStateVec(maxQPowerOcl, usingHostRam);
    BufferPtr nStateBuffer = MakeStateVecBuffer(nStateVec);

    toCopy->clFinish();

    WaitCall(apiCall, ngc, ngs, { stateBuffer, toCopy->stateBuffer, poolItem->ulongBuffer, nStateBuffer });

    stateVec = nStateVec;
    ResetStateBuffer(nStateBuffer);

    SubtractAlloc(sizeof(complex) * oMaxQPower);
}

bitLenInt QEngineCUDA::Compose(QEngineCUDAPtr toCopy)
{
    const bitLenInt result = qubitCount;

    const bitCapIntOcl oQubitCount = toCopy->qubitCount;
    const bitCapIntOcl nQubitCount = qubitCount + oQubitCount;
    const bitCapIntOcl nMaxQPower = pow2Ocl(nQubitCount);
    const bitCapIntOcl startMask = maxQPowerOcl - 1U;
    const bitCapIntOcl endMask = (toCopy->maxQPowerOcl - 1U) << qubitCount;
    const bitCapIntOcl bciArgs[BCI_ARG_LEN]{ nMaxQPower, qubitCount, startMask, endMask, 0U, 0U, 0U, 0U, 0U, 0U };

    OCLAPI api_call;
    if (nMaxQPower <= nrmGroupCount) {
        api_call = OCL_API_COMPOSE_WIDE;
    } else {
        api_call = OCL_API_COMPOSE;
    }

    Compose(api_call, bciArgs, toCopy);

    return result;
}

bitLenInt QEngineCUDA::Compose(QEngineCUDAPtr toCopy, bitLenInt start)
{
    if (start > qubitCount) {
        throw std::invalid_argument("QEngineCUDA::Compose start index is out-of-bounds!");
    }

    const bitLenInt result = start;

    const bitLenInt oQubitCount = toCopy->qubitCount;
    const bitLenInt nQubitCount = qubitCount + oQubitCount;
    const bitCapIntOcl nMaxQPower = pow2Ocl(nQubitCount);
    const bitCapIntOcl startMask = pow2Ocl(start) - 1U;
    const bitCapIntOcl midMask = bitRegMaskOcl(start, oQubitCount);
    const bitCapIntOcl endMask = pow2MaskOcl(qubitCount + oQubitCount) & ~(startMask | midMask);
    const bitCapIntOcl bciArgs[BCI_ARG_LEN]{ nMaxQPower, qubitCount, oQubitCount, startMask, midMask, endMask, start,
        0U, 0U, 0U };

    Compose(OCL_API_COMPOSE_MID, bciArgs, toCopy);

    return result;
}

void QEngineCUDA::DecomposeDispose(bitLenInt start, bitLenInt length, QEngineCUDAPtr destination)
{
    // "Dispose" is basically the same as decompose, except "Dispose" throws the removed bits away.

    if (isBadBitRange(start, length, qubitCount)) {
        throw std::invalid_argument("QEngineCUDA::DecomposeDispose range is out-of-bounds!");
    }

    if (!length) {
        return;
    }

    if (!stateBuffer) {
        SetQubitCount(qubitCount - length);
        if (destination) {
            destination->ZeroAmplitudes();
        }
        return;
    }

    if (destination && !destination->stateBuffer) {
        // Reinitialize stateVec RAM
        destination->SetPermutation(ZERO_BCI);
    }

    if (doNormalize) {
        NormalizeState();
    }
    if (destination && destination->doNormalize) {
        destination->NormalizeState();
    }

    const bitLenInt nLength = qubitCount - length;

    if (!nLength) {
        if (destination != NULL) {
            destination->stateVec = stateVec;
            destination->stateBuffer = stateBuffer;
            stateBuffer = NULL;
            stateVec = NULL;
        }
        SetQubitCount(0U);
        // This will be cleared by the destructor:
        SubtractAlloc(sizeof(complex) * pow2Ocl(qubitCount));
        stateVec = AllocStateVec(maxQPowerOcl, usingHostRam);
        stateBuffer = MakeStateVecBuffer(stateVec);

        return;
    }

    const bitCapIntOcl partPower = pow2Ocl(length);
    const bitCapIntOcl remainderPower = pow2Ocl(nLength);
    const bitCapIntOcl oMaxQPower = maxQPowerOcl;
    bitCapIntOcl bciArgs[BCI_ARG_LEN]{ partPower, remainderPower, start, length, 0U, 0U, 0U, 0U, 0U, 0U };

    const size_t remainderDiff = 2 * sizeof(real1) * remainderPower;
    AddAlloc(remainderDiff);

    // The "remainder" bits will always be maintained.
    BufferPtr probBuffer1 = MakeBuffer(CL_MEM_READ_WRITE, sizeof(real1) * remainderPower);
    ClearBuffer(probBuffer1, 0U, remainderPower >> 1U);
    BufferPtr angleBuffer1 = MakeBuffer(CL_MEM_READ_WRITE, sizeof(real1) * remainderPower);
    ClearBuffer(angleBuffer1, 0U, remainderPower >> 1U);

    // The removed "part" is only necessary for Decompose.
    BufferPtr probBuffer2, angleBuffer2;
    const size_t partDiff = 2 * sizeof(real1) * partPower;
    if (destination) {
        AddAlloc(2 * sizeof(real1) * partPower);
        probBuffer2 = MakeBuffer(CL_MEM_READ_WRITE, sizeof(real1) * partPower);
        ClearBuffer(probBuffer2, 0U, partPower >> 1U);
        angleBuffer2 = MakeBuffer(CL_MEM_READ_WRITE, sizeof(real1) * partPower);
        ClearBuffer(angleBuffer2, 0U, partPower >> 1U);
    }

    PoolItemPtr poolItem = GetFreePoolItem();
    DISPATCH_WRITE(poolItem->ulongBuffer, sizeof(bitCapIntOcl) << 2U, bciArgs);

    const bitCapIntOcl largerPower = partPower > remainderPower ? partPower : remainderPower;

    const size_t ngc = FixWorkItemCount(largerPower, nrmGroupCount);
    const size_t ngs = FixGroupSize(ngc, nrmGroupSize);

    // Call the kernel that calculates bit probability and angle, retaining both parts.
    if (destination) {
        QueueCall(OCL_API_DECOMPOSEPROB, ngc, ngs,
            { stateBuffer, poolItem->ulongBuffer, probBuffer1, angleBuffer1, probBuffer2, angleBuffer2 });
    } else {
        QueueCall(OCL_API_DISPOSEPROB, ngc, ngs, { stateBuffer, poolItem->ulongBuffer, probBuffer1, angleBuffer1 });
    }

    SetQubitCount(nLength);

    // If we Decompose, calculate the state of the bit system removed.
    if (destination) {
        bciArgs[0] = partPower;

        destination->clFinish();

        poolItem = GetFreePoolItem();
        DISPATCH_WRITE(poolItem->ulongBuffer, sizeof(bitCapIntOcl), bciArgs);

        const size_t ngc2 = FixWorkItemCount(partPower, nrmGroupCount);
        const size_t ngs2 = FixGroupSize(ngc2, nrmGroupSize);

        const size_t oNStateVecSize = maxQPowerOcl * sizeof(complex);

        WaitCall(OCL_API_DECOMPOSEAMP, ngc2, ngs2,
            { probBuffer2, angleBuffer2, poolItem->ulongBuffer, destination->stateBuffer });

        probBuffer2.reset();
        angleBuffer2.reset();

        SubtractAlloc(partDiff);

        if (!(destination->useHostRam) && destination->stateVec &&
            oNStateVecSize <= destination->device_context->GetMaxAlloc() &&
            (2 * oNStateVecSize) <= destination->device_context->GetGlobalSize()) {

            BufferPtr nSB = destination->MakeStateVecBuffer(NULL);

            destination->clFinish();
            clFinish();

            tryCuda("Failed to enqueue buffer copy", [&] {
                return hipMemcpy(nSB.get(), destination->stateBuffer.get(), sizeof(complex) * maxQPowerOcl,
                    hipMemcpyDeviceToDevice);
            });

            destination->stateBuffer = nSB;
            destination->stateVec = NULL;
        }
    }

    // If we either Decompose or Dispose, calculate the state of the bit system that remains.
    bciArgs[0] = maxQPowerOcl;
    poolItem = GetFreePoolItem();
    DISPATCH_WRITE(poolItem->ulongBuffer, sizeof(bitCapIntOcl), bciArgs);

    const size_t ngc3 = FixWorkItemCount(maxQPowerOcl, nrmGroupCount);
    const size_t ngs3 = FixGroupSize(ngc, nrmGroupSize);

    if (stateVec && !usingHostRam) {
        FreeStateVec();
    }
    // Drop references to state vector buffer, which we're done with.
    ResetStateBuffer(NULL);
    SubtractAlloc(sizeof(complex) * oMaxQPower);

    std::shared_ptr<complex> nStateVec = AllocStateVec(maxQPowerOcl, usingHostRam);
    BufferPtr nStateBuffer = MakeStateVecBuffer(nStateVec);

    stateVec = nStateVec;
    ResetStateBuffer(nStateBuffer);

    // Tell QueueCall to track deallocation:
    QueueCall(OCL_API_DECOMPOSEAMP, ngc3, ngs3, { probBuffer1, angleBuffer1, poolItem->ulongBuffer, stateBuffer }, 0U,
        remainderDiff);
}

void QEngineCUDA::Decompose(bitLenInt start, QInterfacePtr destination)
{
    DecomposeDispose(start, destination->GetQubitCount(), std::dynamic_pointer_cast<QEngineCUDA>(destination));
}

void QEngineCUDA::Dispose(bitLenInt start, bitLenInt length) { DecomposeDispose(start, length, (QEngineCUDAPtr)NULL); }

void QEngineCUDA::Dispose(bitLenInt start, bitLenInt length, bitCapInt disposedPerm)
{
    if (!length) {
        return;
    }

    if (!stateBuffer) {
        SetQubitCount(qubitCount - length);
        return;
    }

    if (length == qubitCount) {
        // This will be cleared by the destructor:
        stateVec = NULL;
        stateBuffer = NULL;
        SubtractAlloc(sizeof(complex) * pow2Ocl(qubitCount));
        SetQubitCount(0U);
        return;
    }

    if (doNormalize) {
        NormalizeState();
    }

    PoolItemPtr poolItem = GetFreePoolItem();

    const bitLenInt nLength = qubitCount - length;
    const bitCapIntOcl remainderPower = pow2Ocl(nLength);
    const size_t sizeDiff = sizeof(complex) * maxQPowerOcl;
    const bitCapIntOcl skipMask = pow2Ocl(start) - 1U;
    const bitCapIntOcl disposedRes = (disposedPerm << start).bits[0U];

    const bitCapIntOcl bciArgs[BCI_ARG_LEN]{ remainderPower, length, skipMask, disposedRes, 0U, 0U, 0U, 0U, 0U, 0U };

    DISPATCH_WRITE(poolItem->ulongBuffer, sizeof(bitCapIntOcl) << 2U, bciArgs);

    SetQubitCount(nLength);

    const size_t ngc = FixWorkItemCount(maxQPowerOcl, nrmGroupCount);
    const size_t ngs = FixGroupSize(ngc, nrmGroupSize);

    AddAlloc(sizeof(complex) * maxQPowerOcl);
    std::shared_ptr<complex> nStateVec = AllocStateVec(maxQPowerOcl);
    BufferPtr nStateBuffer = MakeStateVecBuffer(nStateVec);

    QueueCall(OCL_API_DISPOSE, ngc, ngs, { stateBuffer, poolItem->ulongBuffer, nStateBuffer });

    stateVec = nStateVec;
    ResetStateBuffer(nStateBuffer);

    SubtractAlloc(sizeDiff);
}

bitLenInt QEngineCUDA::Allocate(bitLenInt start, bitLenInt length)
{
    if (!length) {
        return start;
    }

    QEngineCUDAPtr nQubits = std::make_shared<QEngineCUDA>(length, 0U, rand_generator, ONE_CMPLX, doNormalize,
        randGlobalPhase, useHostRam, deviceID, hardware_rand_generator != NULL, false, (real1_f)amplitudeFloor);
    return Compose(nQubits, start);
}

real1_f QEngineCUDA::Probx(OCLAPI api_call, const bitCapIntOcl* bciArgs)
{
    if (doNormalize) {
        NormalizeState();
    }

    if (!stateBuffer) {
        return ZERO_R1_F;
    }

    PoolItemPtr poolItem = GetFreePoolItem();
    DISPATCH_TEMP_WRITE(poolItem->ulongBuffer, sizeof(bitCapIntOcl) << 2U, bciArgs);

    const bitCapIntOcl maxI = bciArgs[0];
    const size_t ngc = FixWorkItemCount(maxI, nrmGroupCount);
    const size_t ngs = FixGroupSize(ngc, nrmGroupSize);

    QueueCall(api_call, ngc, ngs, { stateBuffer, poolItem->ulongBuffer, nrmBuffer }, sizeof(real1) * ngs);

    real1 oneChance;
    WAIT_REAL1_SUM(nrmBuffer, ngc / ngs, nrmArray, &oneChance);

    return clampProb((real1_f)oneChance);
}

/// PSEUDO-QUANTUM Direct measure of bit probability to be in |1> state
real1_f QEngineCUDA::Prob(bitLenInt qubit)
{
    if (qubit >= qubitCount) {
        throw std::invalid_argument("QEngineCUDA::Prob qubit index parameter must be within allocated qubit bounds!");
    }

    if (qubitCount == 1) {
        return ProbAll(1);
    }

    if (!stateBuffer) {
        return ZERO_R1_F;
    }

    const bitCapIntOcl qPower = pow2Ocl(qubit);
    const bitCapIntOcl bciArgs[BCI_ARG_LEN]{ maxQPowerOcl >> 1U, qPower, 0U, 0U, 0U, 0U, 0U, 0U, 0U, 0U };

    return Probx(OCL_API_PROB, bciArgs);
}

real1_f QEngineCUDA::CtrlOrAntiProb(bool controlState, bitLenInt control, bitLenInt target)
{
    if (!stateBuffer) {
        return ZERO_R1_F;
    }

    real1_f controlProb = Prob(control);
    if (!controlState) {
        controlProb = ONE_R1 - controlProb;
    }
    if (controlProb <= FP_NORM_EPSILON) {
        return ZERO_R1;
    }
    if ((ONE_R1 - controlProb) <= FP_NORM_EPSILON) {
        return Prob(target);
    }

    if (target >= qubitCount) {
        throw std::invalid_argument(
            "QEngineCUDA::CtrlOrAntiProb target index parameter must be within allocated qubit bounds!");
    }

    const bitCapIntOcl qPower = pow2Ocl(target);
    const bitCapIntOcl qControlPower = pow2Ocl(control);
    const bitCapIntOcl qControlMask = controlState ? qControlPower : 0U;
    const bitCapIntOcl bciArgs[BCI_ARG_LEN]{ maxQPowerOcl >> 2U, qPower, qControlPower, qControlMask, 0U, 0U, 0U, 0U,
        0U, 0U };

    real1_f oneChance = Probx(OCL_API_CPROB, bciArgs);
    oneChance /= controlProb;

    return clampProb((real1_f)oneChance);
}

// Returns probability of permutation of the register
real1_f QEngineCUDA::ProbReg(bitLenInt start, bitLenInt length, bitCapInt permutation)
{
    if (!start && qubitCount == length) {
        return ProbAll(permutation);
    }

    const bitCapIntOcl perm = (permutation << start).bits[0U];
    const bitCapIntOcl bciArgs[BCI_ARG_LEN]{ maxQPowerOcl >> length, perm, start, length, 0U, 0U, 0U, 0U, 0U, 0U };

    return Probx(OCL_API_PROBREG, bciArgs);
}

void QEngineCUDA::ProbRegAll(bitLenInt start, bitLenInt length, real1* probsArray)
{
    const bitCapIntOcl lengthPower = pow2Ocl(length);
    const bitCapIntOcl maxJ = maxQPowerOcl >> length;

    if (doNormalize) {
        NormalizeState();
    }

    if (!stateBuffer) {
        std::fill(probsArray, probsArray + lengthPower, ZERO_R1);
        return;
    }

    const bitCapIntOcl bciArgs[BCI_ARG_LEN]{ lengthPower, maxJ, start, length, 0U, 0U, 0U, 0U, 0U, 0U };

    PoolItemPtr poolItem = GetFreePoolItem();

    DISPATCH_WRITE(poolItem->ulongBuffer, sizeof(bitCapIntOcl) << 2U, bciArgs);

    AddAlloc(sizeof(real1) * lengthPower);
    BufferPtr probsBuffer = MakeBuffer(CL_MEM_WRITE_ONLY, sizeof(real1) * lengthPower);

    const size_t ngc = FixWorkItemCount(lengthPower, nrmGroupCount);
    const size_t ngs = FixGroupSize(ngc, nrmGroupSize);

    QueueCall(OCL_API_PROBREGALL, ngc, ngs, { stateBuffer, poolItem->ulongBuffer, probsBuffer });

    DISPATCH_BLOCK_READ(probsBuffer, 0U, sizeof(real1) * lengthPower, probsArray);

    probsBuffer.reset();

    SubtractAlloc(sizeof(real1) * lengthPower);
}

// Returns probability of permutation of the register
real1_f QEngineCUDA::ProbMask(bitCapInt mask, bitCapInt permutation)
{
    if (bi_compare(mask, maxQPowerOcl) >= 0) {
        throw std::invalid_argument("QEngineCUDA::ProbMask mask out-of-bounds!");
    }

    if (doNormalize) {
        NormalizeState();
    }

    if (!stateBuffer) {
        return ZERO_R1_F;
    }

    bitCapIntOcl v = mask.bits[0U]; // count the number of bits set in v
    bitLenInt length; // c accumulates the total bits set in v
    std::vector<bitCapIntOcl> skipPowersVec;
    for (length = 0U; v; ++length) {
        bitCapIntOcl oldV = v;
        v &= v - 1U; // clear the least significant bit set
        skipPowersVec.push_back((v ^ oldV) & oldV);
    }

    const bitCapIntOcl bciArgs[BCI_ARG_LEN]{ maxQPowerOcl >> length, mask.bits[0U], permutation.bits[0U], length, 0U,
        0U, 0U, 0U, 0U, 0U };

    PoolItemPtr poolItem = GetFreePoolItem();

    DISPATCH_WRITE(poolItem->ulongBuffer, sizeof(bitCapIntOcl) << 2U, bciArgs);

    std::unique_ptr<bitCapIntOcl[]> skipPowers(new bitCapIntOcl[length]);
    std::copy(skipPowersVec.begin(), skipPowersVec.end(), skipPowers.get());
    BufferPtr qPowersBuffer =
        MakeBuffer(CL_MEM_COPY_HOST_PTR | CL_MEM_READ_ONLY, sizeof(bitCapIntOcl) * length, skipPowers.get());
    skipPowers.reset();

    const bitCapIntOcl maxI = bciArgs[0];
    const size_t ngc = FixWorkItemCount(maxI, nrmGroupCount);
    const size_t ngs = FixGroupSize(ngc, nrmGroupSize);

    QueueCall(OCL_API_PROBMASK, ngc, ngs, { stateBuffer, poolItem->ulongBuffer, nrmBuffer, qPowersBuffer },
        sizeof(real1) * ngs);

    real1 oneChance;
    WAIT_REAL1_SUM(nrmBuffer, ngc / ngs, nrmArray, &oneChance);

    return clampProb((real1_f)oneChance);
}

void QEngineCUDA::ProbMaskAll(bitCapInt mask, real1* probsArray)
{
    if (bi_compare(mask, maxQPowerOcl) >= 0) {
        throw std::invalid_argument("QEngineCUDA::ProbMaskAll mask out-of-bounds!");
    }

    if (doNormalize) {
        NormalizeState();
    }

    bitCapIntOcl v = mask.bits[0U]; // count the number of bits set in v
    bitLenInt length;
    std::vector<bitCapIntOcl> powersVec;
    for (length = 0U; v; ++length) {
        bitCapIntOcl oldV = v;
        v &= v - 1U; // clear the least significant bit set
        powersVec.push_back((v ^ oldV) & oldV);
    }

    const bitCapIntOcl lengthPower = pow2Ocl(length);
    const bitCapIntOcl maxJ = maxQPowerOcl >> length;

    if (!stateBuffer) {
        std::fill(probsArray, probsArray + lengthPower, ZERO_R1);
        return;
    }

    if ((lengthPower * lengthPower) < nrmGroupCount) {
        // With "lengthPower" count of threads, compared to a redundancy of "lengthPower" with full utilization, this is
        // close to the point where it becomes more efficient to rely on iterating through ProbReg calls.
        QEngine::ProbMaskAll(mask, probsArray);
        return;
    }

    v = ~mask.bits[0U] & (maxQPowerOcl - 1U); // count the number of bits set in v
    bitCapIntOcl skipPower;
    bitLenInt skipLength = 0U; // c accumulates the total bits set in v
    std::vector<bitCapIntOcl> skipPowersVec;
    for (skipLength = 0U; v; ++skipLength) {
        bitCapIntOcl oldV = v;
        v &= v - 1U; // clear the least significant bit set
        skipPower = (v ^ oldV) & oldV;
        skipPowersVec.push_back(skipPower);
    }

    const bitCapIntOcl bciArgs[BCI_ARG_LEN]{ lengthPower, maxJ, length, skipLength, 0U, 0U, 0U, 0U, 0U, 0U };

    PoolItemPtr poolItem = GetFreePoolItem();

    DISPATCH_WRITE(poolItem->ulongBuffer, sizeof(bitCapIntOcl) << 2U, bciArgs);

    size_t sizeDiff = sizeof(real1) * lengthPower + sizeof(bitCapIntOcl) * length + sizeof(bitCapIntOcl) * skipLength;
    AddAlloc(sizeDiff);

    BufferPtr probsBuffer = MakeBuffer(CL_MEM_WRITE_ONLY, sizeof(real1) * lengthPower);

    std::unique_ptr<bitCapIntOcl[]> powers(new bitCapIntOcl[length]);
    std::copy(powersVec.begin(), powersVec.end(), powers.get());
    BufferPtr qPowersBuffer =
        MakeBuffer(CL_MEM_COPY_HOST_PTR | CL_MEM_READ_ONLY, sizeof(bitCapIntOcl) * length, powers.get());
    powers.reset();

    std::unique_ptr<bitCapIntOcl[]> skipPowers(new bitCapIntOcl[skipLength]);
    std::copy(skipPowersVec.begin(), skipPowersVec.end(), skipPowers.get());
    BufferPtr qSkipPowersBuffer =
        MakeBuffer(CL_MEM_COPY_HOST_PTR | CL_MEM_READ_ONLY, sizeof(bitCapIntOcl) * skipLength, skipPowers.get());
    skipPowers.reset();

    const size_t ngc = FixWorkItemCount(lengthPower, nrmGroupCount);
    const size_t ngs = FixGroupSize(ngc, nrmGroupSize);

    QueueCall(OCL_API_PROBMASKALL, ngc, ngs,
        { stateBuffer, poolItem->ulongBuffer, probsBuffer, qPowersBuffer, qSkipPowersBuffer });

    DISPATCH_BLOCK_READ(probsBuffer, 0U, sizeof(real1) * lengthPower, probsArray);

    probsBuffer.reset();
    qPowersBuffer.reset();
    qSkipPowersBuffer.reset();

    SubtractAlloc(sizeDiff);
}

real1_f QEngineCUDA::ProbParity(bitCapInt mask)
{
    if (bi_compare(mask, maxQPowerOcl) >= 0) {
        throw std::invalid_argument("QEngineCUDA::ProbParity mask out-of-bounds!");
    }

    // If no bits in mask:
    if (bi_compare_0(mask) == 0) {
        return ZERO_R1_F;
    }

    // If only one bit in mask:
    if (isPowerOfTwo(mask)) {
        return Prob(log2(mask));
    }

    const bitCapIntOcl bciArgs[BCI_ARG_LEN]{ maxQPowerOcl, mask.bits[0U], 0U, 0U, 0U, 0U, 0U, 0U, 0U, 0U };

    return Probx(OCL_API_PROBPARITY, bciArgs);
}

bool QEngineCUDA::ForceMParity(bitCapInt mask, bool result, bool doForce)
{
    if (bi_compare(mask, maxQPowerOcl) >= 0) {
        throw std::invalid_argument("QEngineCUDA::ForceMParity mask out-of-bounds!");
    }

    if (!stateBuffer || (bi_compare_0(mask) == 0)) {
        return false;
    }

    // If only one bit in mask:
    if (isPowerOfTwo(mask)) {
        return ForceM(log2(mask), result, doForce);
    }

    if (!doForce) {
        result = (Rand() <= ProbParity(mask));
    }

    const bitCapIntOcl bciArgs[BCI_ARG_LEN]{ maxQPowerOcl, mask.bits[0U], result ? 1U : 0U, 0U, 0U, 0U, 0U, 0U, 0U,
        0U };

    runningNorm = Probx(OCL_API_FORCEMPARITY, bciArgs);

    if (!doNormalize) {
        NormalizeState();
    }

    return result;
}

real1_f QEngineCUDA::ExpectationBitsAll(const std::vector<bitLenInt>& bits, bitCapInt offset)
{
    if (bits.size() == 1U) {
        return Prob(bits[0]);
    }

    if (!stateBuffer || !bits.size()) {
        return ZERO_R1_F;
    }

    if (doNormalize) {
        NormalizeState();
    }

    std::unique_ptr<bitCapIntOcl[]> bitPowers(new bitCapIntOcl[bits.size()]);
    for (bitLenInt p = 0U; p < bits.size(); ++p) {
        bitPowers[p] = pow2Ocl(bits[p]);
    }

    PoolItemPtr poolItem = GetFreePoolItem();

    BufferPtr bitMapBuffer = MakeBuffer(CL_MEM_READ_ONLY, sizeof(bitCapIntOcl) * bits.size());
    DISPATCH_WRITE(bitMapBuffer, sizeof(bitCapIntOcl) * bits.size(), bitPowers.get());
    const bitCapIntOcl bciArgs[BCI_ARG_LEN]{ maxQPowerOcl, (bitCapIntOcl)bits.size(), offset.bits[0U], 0U, 0U, 0U, 0U,
        0U, 0U, 0U };
    DISPATCH_WRITE(poolItem->ulongBuffer, sizeof(bitCapIntOcl) * 3, bciArgs);

    const size_t ngc = FixWorkItemCount(maxQPowerOcl, nrmGroupCount);
    const size_t ngs = FixGroupSize(ngc, nrmGroupSize);

    QueueCall(OCL_API_EXPPERM, ngc, ngs, { stateBuffer, poolItem->ulongBuffer, bitMapBuffer, nrmBuffer },
        sizeof(real1) * ngs);

    real1_f expectation;
    WAIT_REAL1_SUM(nrmBuffer, ngc / ngs, nrmArray, &expectation);

    return expectation;
}

real1_f QEngineCUDA::GetExpectation(bitLenInt valueStart, bitLenInt valueLength)
{
    real1 average = ZERO_R1;
    real1 totProb = ZERO_R1;
    const bitCapIntOcl outputMask = bitRegMaskOcl(valueStart, valueLength);
    LockSync(CL_MAP_READ);
    for (bitCapIntOcl i = 0U; i < maxQPower; ++i) {
        const bitCapIntOcl outputInt = (i & outputMask) >> valueStart;
        const real1 prob = norm(stateVec.get()[i]);
        totProb += prob;
        average += prob * outputInt;
    }
    UnlockSync();
    if (totProb > ZERO_R1) {
        average /= totProb;
    }

    return (real1_f)average;
}

void QEngineCUDA::ArithmeticCall(
    OCLAPI api_call, const bitCapIntOcl (&bciArgs)[BCI_ARG_LEN], const unsigned char* values, bitCapIntOcl valuesPower)
{
    CArithmeticCall(api_call, bciArgs, NULL, 0U, values, valuesPower);
}
void QEngineCUDA::CArithmeticCall(OCLAPI api_call, const bitCapIntOcl (&bciArgs)[BCI_ARG_LEN],
    bitCapIntOcl* controlPowers, bitLenInt controlLen, const unsigned char* values, bitCapIntOcl valuesPower)
{
    CHECK_ZERO_SKIP();

    size_t sizeDiff = sizeof(complex) * maxQPowerOcl;
    if (controlLen) {
        sizeDiff += sizeof(bitCapIntOcl) * controlLen;
    }
    if (values) {
        sizeDiff += sizeof(unsigned char) * valuesPower;
    }
    AddAlloc(sizeDiff);

    // Allocate a temporary nStateVec, or use the one supplied.
    std::shared_ptr<complex> nStateVec = AllocStateVec(maxQPowerOcl);
    BufferPtr nStateBuffer;
    BufferPtr controlBuffer;
    if (controlLen) {
        controlBuffer =
            MakeBuffer(CL_MEM_COPY_HOST_PTR | CL_MEM_READ_ONLY, sizeof(bitCapIntOcl) * controlLen, controlPowers);
    }

    nStateBuffer = MakeStateVecBuffer(nStateVec);

    if (controlLen) {
        tryCuda("Failed to enqueue buffer copy", [&] {
            return hipMemcpy(
                nStateBuffer.get(), stateBuffer.get(), sizeof(complex) * maxQPowerOcl, hipMemcpyDeviceToDevice);
        });
    } else {
        ClearBuffer(nStateBuffer, 0U, maxQPowerOcl);
    }

    PoolItemPtr poolItem = GetFreePoolItem();
    DISPATCH_WRITE(poolItem->ulongBuffer, sizeof(bitCapIntOcl) * BCI_ARG_LEN, bciArgs);

    const bitCapIntOcl maxI = bciArgs[0];
    const size_t ngc = FixWorkItemCount(maxI, nrmGroupCount);
    const size_t ngs = FixGroupSize(ngc, nrmGroupSize);

    std::vector<BufferPtr> oclArgs = { stateBuffer, poolItem->ulongBuffer, nStateBuffer };

    BufferPtr loadBuffer;
    if (values) {
        loadBuffer =
            MakeBuffer(CL_MEM_COPY_HOST_PTR | CL_MEM_READ_ONLY, sizeof(unsigned char) * valuesPower, (void*)values);
        oclArgs.push_back(loadBuffer);
    }
    if (controlLen) {
        oclArgs.push_back(controlBuffer);
    }

    QueueCall(api_call, ngc, ngs, oclArgs);

    stateVec = nStateVec;
    ResetStateBuffer(nStateBuffer);

    SubtractAlloc(sizeDiff);
}

void QEngineCUDA::ROx(OCLAPI api_call, bitLenInt shift, bitLenInt start, bitLenInt length)
{
    if (isBadBitRange(start, length, qubitCount)) {
        throw std::invalid_argument("QEngineCUDA::ROx range is out-of-bounds!");
    }

    if (!length) {
        return;
    }

    shift %= length;
    if (!shift) {
        return;
    }

    const bitCapIntOcl lengthPower = pow2Ocl(length);
    const bitCapIntOcl regMask = (lengthPower - 1U) << start;
    const bitCapIntOcl otherMask = (maxQPowerOcl - 1U) & (~regMask);
    const bitCapIntOcl bciArgs[BCI_ARG_LEN]{ maxQPowerOcl, regMask, otherMask, lengthPower, start, shift, length, 0U,
        0U, 0U };

    ArithmeticCall(api_call, bciArgs);
}

/// "Circular shift left" - shift bits left, and carry last bits.
void QEngineCUDA::ROL(bitLenInt shift, bitLenInt start, bitLenInt length) { ROx(OCL_API_ROL, shift, start, length); }

#if ENABLE_ALU
/// Add or Subtract integer (without sign or carry)
void QEngineCUDA::INT(OCLAPI api_call, bitCapIntOcl toMod, bitLenInt start, bitLenInt length)
{
    if (isBadBitRange(start, length, qubitCount)) {
        throw std::invalid_argument("QEngineCUDA::INT range is out-of-bounds!");
    }

    if (!length) {
        return;
    }

    const bitCapIntOcl lengthPower = pow2Ocl(length);
    const bitCapIntOcl lengthMask = lengthPower - 1U;
    toMod &= lengthMask;
    if (!toMod) {
        return;
    }

    const bitCapIntOcl regMask = lengthMask << start;
    const bitCapIntOcl otherMask = (maxQPowerOcl - 1U) & ~(regMask);
    const bitCapIntOcl bciArgs[BCI_ARG_LEN]{ maxQPowerOcl, regMask, otherMask, lengthPower, start, toMod, 0U, 0U, 0U,
        0U };

    ArithmeticCall(api_call, bciArgs);
}

/// Add or Subtract integer (without sign or carry, with controls)
void QEngineCUDA::CINT(
    OCLAPI api_call, bitCapIntOcl toMod, bitLenInt start, bitLenInt length, const std::vector<bitLenInt>& controls)
{
    if (isBadBitRange(start, length, qubitCount)) {
        throw std::invalid_argument("QEngineCUDA::CINT range is out-of-bounds!");
    }

    ThrowIfQbIdArrayIsBad(controls, qubitCount, "QEngineCUDA::CINT control is out-of-bounds!");

    if (!length) {
        return;
    }

    const bitCapIntOcl lengthPower = pow2Ocl(length);
    const bitCapIntOcl lengthMask = lengthPower - 1U;
    toMod &= lengthMask;
    if (!toMod) {
        return;
    }

    const bitCapIntOcl regMask = lengthMask << start;

    bitCapIntOcl controlMask = 0U;
    std::unique_ptr<bitCapIntOcl[]> controlPowers(new bitCapIntOcl[controls.size()]);
    for (bitLenInt i = 0U; i < controls.size(); ++i) {
        controlPowers[i] = pow2Ocl(controls[i]);
        controlMask |= controlPowers[i];
    }
    std::sort(controlPowers.get(), controlPowers.get() + controls.size());

    const bitCapIntOcl otherMask = (maxQPowerOcl - 1U) ^ (regMask | controlMask);
    const bitCapIntOcl bciArgs[BCI_ARG_LEN]{ maxQPowerOcl >> (bitLenInt)controls.size(), regMask, otherMask,
        lengthPower, start, toMod, (bitCapIntOcl)controls.size(), controlMask, 0U, 0U };

    CArithmeticCall(api_call, bciArgs, controlPowers.get(), controls.size());
}

/** Increment integer (without sign, with carry) */
void QEngineCUDA::INC(bitCapInt toAdd, bitLenInt start, bitLenInt length)
{
    INT(OCL_API_INC, toAdd.bits[0U], start, length);
}

void QEngineCUDA::CINC(bitCapInt toAdd, bitLenInt inOutStart, bitLenInt length, const std::vector<bitLenInt>& controls)
{
    if (!controls.size()) {
        INC(toAdd, inOutStart, length);
        return;
    }

    CINT(OCL_API_CINC, toAdd.bits[0U], inOutStart, length, controls);
}

/// Add or Subtract integer (without sign, with carry)
void QEngineCUDA::INTC(OCLAPI api_call, bitCapIntOcl toMod, bitLenInt start, bitLenInt length, bitLenInt carryIndex)
{
    if (isBadBitRange(start, length, qubitCount)) {
        throw std::invalid_argument("QEngineCUDA::INTC range is out-of-bounds!");
    }

    if (carryIndex >= qubitCount) {
        throw std::invalid_argument("QEngineCUDA::INTC carryIndex is out-of-bounds!");
    }

    if (!length) {
        return;
    }

    const bitCapIntOcl lengthPower = pow2Ocl(length);
    const bitCapIntOcl lengthMask = lengthPower - 1U;
    toMod &= lengthMask;
    if (!toMod) {
        return;
    }

    const bitCapIntOcl carryMask = pow2Ocl(carryIndex);
    const bitCapIntOcl regMask = (lengthPower - 1U) << start;
    const bitCapIntOcl otherMask = (maxQPowerOcl - 1U) & (~(regMask | carryMask));
    const bitCapIntOcl bciArgs[BCI_ARG_LEN]{ maxQPowerOcl >> 1U, regMask, otherMask, lengthPower, carryMask, start,
        toMod, 0U, 0U, 0U };

    ArithmeticCall(api_call, bciArgs);
}

/// Common driver method behing INCC and DECC
void QEngineCUDA::INCDECC(bitCapInt toMod, bitLenInt inOutStart, bitLenInt length, bitLenInt carryIndex)
{
    INTC(OCL_API_INCDECC, toMod.bits[0U], inOutStart, length, carryIndex);
}

/// Add or Subtract integer (with overflow, without carry)
void QEngineCUDA::INTS(OCLAPI api_call, bitCapIntOcl toMod, bitLenInt start, bitLenInt length, bitLenInt overflowIndex)
{
    if (isBadBitRange(start, length, qubitCount)) {
        throw std::invalid_argument("QEngineCUDA::INTS range is out-of-bounds!");
    }

    if (overflowIndex >= qubitCount) {
        throw std::invalid_argument("QEngineCUDA::INTS overflowIndex is out-of-bounds!");
    }

    if (!length) {
        return;
    }

    const bitCapIntOcl lengthPower = pow2Ocl(length);
    const bitCapIntOcl lengthMask = lengthPower - 1U;
    toMod &= lengthMask;
    if (!toMod) {
        return;
    }

    const bitCapIntOcl overflowMask = pow2Ocl(overflowIndex);
    const bitCapIntOcl regMask = lengthMask << start;
    const bitCapIntOcl otherMask = (maxQPowerOcl - 1U) ^ regMask;
    const bitCapIntOcl bciArgs[BCI_ARG_LEN]{ maxQPowerOcl, regMask, otherMask, lengthPower, overflowMask, start, toMod,
        0U, 0U, 0U };

    ArithmeticCall(api_call, bciArgs);
}

/** Increment integer (without sign, with carry) */
void QEngineCUDA::INCS(bitCapInt toAdd, bitLenInt start, bitLenInt length, bitLenInt overflowIndex)
{
    INTS(OCL_API_INCS, toAdd.bits[0U], start, length, overflowIndex);
}

/// Add or Subtract integer (with sign, with carry)
void QEngineCUDA::INTSC(OCLAPI api_call, bitCapIntOcl toMod, bitLenInt start, bitLenInt length, bitLenInt overflowIndex,
    bitLenInt carryIndex)
{
    if (isBadBitRange(start, length, qubitCount)) {
        throw std::invalid_argument("QEngineCUDA::INTSC range is out-of-bounds!");
    }

    if (overflowIndex >= qubitCount) {
        throw std::invalid_argument("QEngineCUDA::INTSC overflowIndex is out-of-bounds!");
    }

    if (carryIndex >= qubitCount) {
        throw std::invalid_argument("QEngineCUDA::INTSC carryIndex is out-of-bounds!");
    }

    if (!length) {
        return;
    }

    const bitCapIntOcl lengthPower = pow2Ocl(length);
    const bitCapIntOcl lengthMask = lengthPower - 1U;
    toMod &= lengthMask;
    if (!toMod) {
        return;
    }

    const bitCapIntOcl overflowMask = pow2Ocl(overflowIndex);
    const bitCapIntOcl carryMask = pow2Ocl(carryIndex);
    const bitCapIntOcl inOutMask = lengthMask << start;
    const bitCapIntOcl otherMask = (maxQPowerOcl - 1U) ^ (inOutMask | carryMask);
    const bitCapIntOcl bciArgs[BCI_ARG_LEN]{ maxQPowerOcl >> 1U, inOutMask, otherMask, lengthPower, overflowMask,
        carryMask, start, toMod, 0U, 0U };

    ArithmeticCall(api_call, bciArgs);
}

/** Increment integer (with sign, with carry) */
void QEngineCUDA::INCDECSC(
    bitCapInt toAdd, bitLenInt start, bitLenInt length, bitLenInt overflowIndex, bitLenInt carryIndex)
{
    INTSC(OCL_API_INCDECSC_1, toAdd.bits[0U], start, length, overflowIndex, carryIndex);
}

/// Add or Subtract integer (with sign, with carry)
void QEngineCUDA::INTSC(OCLAPI api_call, bitCapIntOcl toMod, bitLenInt start, bitLenInt length, bitLenInt carryIndex)
{
    if (isBadBitRange(start, length, qubitCount)) {
        throw std::invalid_argument("QEngineCUDA::INTSC range is out-of-bounds!");
    }

    if (carryIndex >= qubitCount) {
        throw std::invalid_argument("QEngineCUDA::INTSC carryIndex is out-of-bounds!");
    }

    const bitCapIntOcl carryMask = pow2Ocl(carryIndex);
    const bitCapIntOcl lengthPower = pow2Ocl(length);
    const bitCapIntOcl inOutMask = (lengthPower - 1U) << start;
    const bitCapIntOcl otherMask = pow2MaskOcl(qubitCount) ^ (inOutMask | carryMask);
    const bitCapIntOcl bciArgs[BCI_ARG_LEN]{ maxQPowerOcl >> 1U, inOutMask, otherMask, lengthPower, carryMask, start,
        toMod, 0U, 0U, 0U };

    ArithmeticCall(api_call, bciArgs);
}

/** Increment integer (with sign, with carry) */
void QEngineCUDA::INCDECSC(bitCapInt toAdd, bitLenInt start, bitLenInt length, bitLenInt carryIndex)
{
    INTSC(OCL_API_INCDECSC_2, toAdd.bits[0U], start, length, carryIndex);
}

#if ENABLE_BCD
/// Add or Subtract integer (BCD)
void QEngineCUDA::INTBCD(OCLAPI api_call, bitCapIntOcl toMod, bitLenInt start, bitLenInt length)
{
    if (isBadBitRange(start, length, qubitCount)) {
        throw std::invalid_argument("QEngineCUDA::INTBCD range is out-of-bounds!");
    }

    if (!length) {
        return;
    }

    const bitLenInt nibbleCount = length / 4;
    if ((nibbleCount << 2U) != length) {
        throw std::invalid_argument("BCD word bit length must be a multiple of 4.");
    }

    const bitCapIntOcl maxPow = intPowOcl(10U, nibbleCount);
    toMod %= maxPow;
    if (!toMod) {
        return;
    }

    const bitCapIntOcl inOutMask = bitRegMaskOcl(start, length);
    const bitCapIntOcl otherMask = (maxQPowerOcl - 1U) ^ inOutMask;
    const bitCapIntOcl bciArgs[BCI_ARG_LEN]{ maxQPowerOcl, inOutMask, otherMask, start, toMod, nibbleCount, 0U, 0U, 0U,
        0U };

    ArithmeticCall(api_call, bciArgs);
}

/** Increment integer (BCD) */
void QEngineCUDA::INCBCD(bitCapInt toAdd, bitLenInt start, bitLenInt length)
{
    INTBCD(OCL_API_INCBCD, toAdd.bits[0U], start, length);
}

/// Add or Subtract integer (BCD, with carry)
void QEngineCUDA::INTBCDC(OCLAPI api_call, bitCapIntOcl toMod, bitLenInt start, bitLenInt length, bitLenInt carryIndex)
{
    if (isBadBitRange(start, length, qubitCount)) {
        throw std::invalid_argument("QEngineCUDA::INTBCDC range is out-of-bounds!");
    }

    if (carryIndex >= qubitCount) {
        throw std::invalid_argument("QEngineCUDA::INTBCDC carryIndex is out-of-bounds!");
    }

    if (!length) {
        return;
    }

    const bitLenInt nibbleCount = length / 4;
    if ((nibbleCount << 2U) != length) {
        throw std::invalid_argument("BCD word bit length must be a multiple of 4.");
    }

    const bitCapIntOcl maxPow = intPowOcl(10U, nibbleCount);
    toMod %= maxPow;
    if (!toMod) {
        return;
    }

    const bitCapIntOcl inOutMask = bitRegMaskOcl(start, length);
    const bitCapIntOcl carryMask = pow2Ocl(carryIndex);
    const bitCapIntOcl otherMask = (maxQPowerOcl - 1U) ^ (inOutMask | carryMask);
    const bitCapIntOcl bciArgs[BCI_ARG_LEN]{ maxQPowerOcl >> 1U, inOutMask, otherMask, carryMask, start, toMod,
        nibbleCount, 0U, 0U, 0U };

    ArithmeticCall(api_call, bciArgs);
}

/** Increment integer (BCD, with carry) */
void QEngineCUDA::INCDECBCDC(bitCapInt toAdd, bitLenInt start, bitLenInt length, bitLenInt carryIndex)
{
    INTBCDC(OCL_API_INCDECBCDC, toAdd.bits[0U], start, length, carryIndex);
}
#endif

/** Multiply by integer */
void QEngineCUDA::MUL(bitCapInt toMul, bitLenInt inOutStart, bitLenInt carryStart, bitLenInt length)
{
    CHECK_ZERO_SKIP();

    SetReg(carryStart, length, ZERO_BCI);

    const bitCapIntOcl lowPower = pow2Ocl(length);
    const bitCapIntOcl toMulOcl = toMul.bits[0U] & (lowPower - 1U);
    if (!toMulOcl) {
        SetReg(inOutStart, length, ZERO_BCI);
        return;
    }

    MULx(OCL_API_MUL, toMulOcl, inOutStart, carryStart, length);
}

/** Divide by integer */
void QEngineCUDA::DIV(bitCapInt toDiv, bitLenInt inOutStart, bitLenInt carryStart, bitLenInt length)
{
    if (bi_compare_0(toDiv) == 0) {
        throw std::runtime_error("DIV by zero");
    }

    MULx(OCL_API_DIV, toDiv.bits[0U], inOutStart, carryStart, length);
}

/** Multiplication modulo N by integer, (out of place) */
void QEngineCUDA::MULModNOut(bitCapInt toMul, bitCapInt modN, bitLenInt inStart, bitLenInt outStart, bitLenInt length)
{
    CHECK_ZERO_SKIP();

    SetReg(outStart, length, ZERO_BCI);

    MULModx(OCL_API_MULMODN_OUT, toMul.bits[0U], modN.bits[0U], inStart, outStart, length);
}

void QEngineCUDA::IMULModNOut(bitCapInt toMul, bitCapInt modN, bitLenInt inStart, bitLenInt outStart, bitLenInt length)
{
    CHECK_ZERO_SKIP();

    MULModx(OCL_API_IMULMODN_OUT, toMul.bits[0U], modN.bits[0U], inStart, outStart, length);
}

/** Raise a classical base to a quantum power, modulo N, (out of place) */
void QEngineCUDA::POWModNOut(bitCapInt base, bitCapInt modN, bitLenInt inStart, bitLenInt outStart, bitLenInt length)
{
    CHECK_ZERO_SKIP();

    if (bi_compare_1(base) == 0) {
        SetReg(outStart, length, ONE_BCI);
        return;
    }

    MULModx(OCL_API_POWMODN_OUT, base.bits[0U], modN.bits[0U], inStart, outStart, length);
}

/** Quantum analog of classical "Full Adder" gate */
void QEngineCUDA::FullAdd(bitLenInt inputBit1, bitLenInt inputBit2, bitLenInt carryInSumOut, bitLenInt carryOut)
{
    FullAdx(inputBit1, inputBit2, carryInSumOut, carryOut, OCL_API_FULLADD);
}

/** Inverse of FullAdd */
void QEngineCUDA::IFullAdd(bitLenInt inputBit1, bitLenInt inputBit2, bitLenInt carryInSumOut, bitLenInt carryOut)
{
    FullAdx(inputBit1, inputBit2, carryInSumOut, carryOut, OCL_API_IFULLADD);
}

void QEngineCUDA::FullAdx(
    bitLenInt inputBit1, bitLenInt inputBit2, bitLenInt carryInSumOut, bitLenInt carryOut, OCLAPI api_call)
{
    CHECK_ZERO_SKIP();

    const bitCapIntOcl bciArgs[BCI_ARG_LEN]{ maxQPowerOcl >> 2U, pow2Ocl(inputBit1), pow2Ocl(inputBit2),
        pow2Ocl(carryInSumOut), pow2Ocl(carryOut), 0U, 0U, 0U, 0U, 0U };

    PoolItemPtr poolItem = GetFreePoolItem();

    DISPATCH_TEMP_WRITE(poolItem->ulongBuffer, sizeof(bitCapIntOcl) * 5, bciArgs);

    const size_t ngc = FixWorkItemCount(bciArgs[0], nrmGroupCount);
    const size_t ngs = FixGroupSize(ngc, nrmGroupSize);

    QueueCall(api_call, ngc, ngs, { stateBuffer, poolItem->ulongBuffer });
}

/** Controlled multiplication by integer */
void QEngineCUDA::CMUL(bitCapInt toMul, bitLenInt inOutStart, bitLenInt carryStart, bitLenInt length,
    const std::vector<bitLenInt>& controls)
{
    CHECK_ZERO_SKIP();

    if (!controls.size()) {
        MUL(toMul, inOutStart, carryStart, length);
        return;
    }

    SetReg(carryStart, length, ZERO_BCI);

    const bitCapIntOcl lowPower = pow2Ocl(length);
    const bitCapIntOcl toMulOcl = toMul.bits[0U] & (lowPower - 1U);
    if (toMulOcl == 1) {
        return;
    }

    CMULx(OCL_API_CMUL, toMulOcl, inOutStart, carryStart, length, controls);
}

/** Controlled division by integer */
void QEngineCUDA::CDIV(bitCapInt toDiv, bitLenInt inOutStart, bitLenInt carryStart, bitLenInt length,
    const std::vector<bitLenInt>& controls)
{
    if (!controls.size()) {
        DIV(toDiv, inOutStart, carryStart, length);
        return;
    }

    if (bi_compare_0(toDiv) == 0) {
        throw std::runtime_error("DIV by zero");
    }

    if (bi_compare_1(toDiv) == 0) {
        return;
    }

    CMULx(OCL_API_CDIV, toDiv.bits[0U], inOutStart, carryStart, length, controls);
}

/** Controlled multiplication modulo N by integer, (out of place) */
void QEngineCUDA::CMULModNOut(bitCapInt toMul, bitCapInt modN, bitLenInt inStart, bitLenInt outStart, bitLenInt length,
    const std::vector<bitLenInt>& controls)
{
    CHECK_ZERO_SKIP();

    if (!controls.size()) {
        MULModNOut(toMul, modN, inStart, outStart, length);
        return;
    }

    SetReg(outStart, length, ZERO_BCI);

    const bitCapIntOcl lowPower = pow2Ocl(length);
    const bitCapIntOcl toMulOcl = toMul.bits[0U] & (lowPower - 1U);
    if (!toMulOcl) {
        return;
    }

    CMULModx(OCL_API_CMULMODN_OUT, toMulOcl, modN.bits[0U], inStart, outStart, length, controls);
}

void QEngineCUDA::CIMULModNOut(bitCapInt toMul, bitCapInt modN, bitLenInt inStart, bitLenInt outStart, bitLenInt length,
    const std::vector<bitLenInt>& controls)
{
    if (!controls.size()) {
        IMULModNOut(toMul, modN, inStart, outStart, length);
        return;
    }

    const bitCapIntOcl lowPower = pow2Ocl(length);
    const bitCapIntOcl toMulOcl = toMul.bits[0U] & (lowPower - 1U);
    if (!toMulOcl) {
        return;
    }

    CMULModx(OCL_API_CIMULMODN_OUT, toMulOcl, modN.bits[0U], inStart, outStart, length, controls);
}

/** Controlled multiplication modulo N by integer, (out of place) */
void QEngineCUDA::CPOWModNOut(bitCapInt base, bitCapInt modN, bitLenInt inStart, bitLenInt outStart, bitLenInt length,
    const std::vector<bitLenInt>& controls)
{
    CHECK_ZERO_SKIP();

    if (!controls.size()) {
        POWModNOut(base, modN, inStart, outStart, length);
        return;
    }

    SetReg(outStart, length, ZERO_BCI);

    CMULModx(OCL_API_CPOWMODN_OUT, base.bits[0U], modN.bits[0U], inStart, outStart, length, controls);
}

void QEngineCUDA::xMULx(OCLAPI api_call, const bitCapIntOcl* bciArgs, BufferPtr controlBuffer)
{
    CHECK_ZERO_SKIP();

    /* Allocate a temporary nStateVec, or use the one supplied. */
    std::shared_ptr<complex> nStateVec = AllocStateVec(maxQPowerOcl);
    BufferPtr nStateBuffer = MakeStateVecBuffer(nStateVec);

    ClearBuffer(nStateBuffer, 0U, maxQPowerOcl);

    PoolItemPtr poolItem = GetFreePoolItem();
    DISPATCH_WRITE(poolItem->ulongBuffer, sizeof(bitCapIntOcl) * 10U, bciArgs);

    const size_t ngc = FixWorkItemCount(bciArgs[0], nrmGroupCount);
    const size_t ngs = FixGroupSize(ngc, nrmGroupSize);

    if (controlBuffer) {
        QueueCall(api_call, ngc, ngs, { stateBuffer, poolItem->ulongBuffer, nStateBuffer, controlBuffer });
    } else {
        QueueCall(api_call, ngc, ngs, { stateBuffer, poolItem->ulongBuffer, nStateBuffer });
    }

    stateVec = nStateVec;
    ResetStateBuffer(nStateBuffer);
}

void QEngineCUDA::MULx(
    OCLAPI api_call, bitCapIntOcl toMod, bitLenInt inOutStart, bitLenInt carryStart, bitLenInt length)
{
    if (isBadBitRange(inOutStart, length, qubitCount)) {
        throw std::invalid_argument("QEngineCUDA::MULx range is out-of-bounds!");
    }

    if (isBadBitRange(carryStart, length, qubitCount)) {
        throw std::invalid_argument("QEngineCUDA::MULx range is out-of-bounds!");
    }

    const bitCapIntOcl lowMask = pow2MaskOcl(length);
    const bitCapIntOcl inOutMask = lowMask << inOutStart;
    const bitCapIntOcl carryMask = lowMask << carryStart;
    const bitCapIntOcl skipMask = pow2MaskOcl(carryStart);
    const bitCapIntOcl otherMask = (maxQPowerOcl - 1U) ^ (inOutMask | carryMask);
    const bitCapIntOcl bciArgs[BCI_ARG_LEN]{ maxQPowerOcl >> length, toMod, inOutMask, carryMask, otherMask, length,
        inOutStart, carryStart, skipMask, 0U };

    xMULx(api_call, bciArgs, NULL);
}

void QEngineCUDA::MULModx(
    OCLAPI api_call, bitCapIntOcl toMod, bitCapIntOcl modN, bitLenInt inStart, bitLenInt outStart, bitLenInt length)
{
    if (isBadBitRange(inStart, length, qubitCount)) {
        throw std::invalid_argument("QEngineCUDA::MULModx range is out-of-bounds!");
    }

    if (isBadBitRange(outStart, length, qubitCount)) {
        throw std::invalid_argument("QEngineCUDA::MULModx range is out-of-bounds!");
    }

    if (!toMod) {
        return;
    }

    const bitCapIntOcl lowMask = pow2MaskOcl(length);
    const bitCapIntOcl inMask = lowMask << inStart;
    const bitCapIntOcl modMask = (isPowerOfTwo(modN) ? modN : pow2Ocl(log2(modN) + 1U)) - 1U;
    const bitCapIntOcl outMask = modMask << outStart;
    const bitCapIntOcl skipMask = pow2MaskOcl(outStart);
    const bitCapIntOcl otherMask = (maxQPowerOcl - 1U) ^ (inMask | outMask);
    const bitCapIntOcl bciArgs[BCI_ARG_LEN]{ maxQPowerOcl >> length, toMod, inMask, outMask, otherMask, length, inStart,
        outStart, skipMask, modN };

    xMULx(api_call, bciArgs, NULL);
}

void QEngineCUDA::CMULx(OCLAPI api_call, bitCapIntOcl toMod, bitLenInt inOutStart, bitLenInt carryStart,
    bitLenInt length, const std::vector<bitLenInt>& controls)
{
    if (isBadBitRange(inOutStart, length, qubitCount)) {
        throw std::invalid_argument("QEngineCUDA::CMULx range is out-of-bounds!");
    }

    if (isBadBitRange(carryStart, length, qubitCount)) {
        throw std::invalid_argument("QEngineCUDA::CMULx range is out-of-bounds!");
    }

    ThrowIfQbIdArrayIsBad(controls, qubitCount, "QEngineCUDA::CMULx control is out-of-bounds!");

    const bitCapIntOcl lowMask = pow2MaskOcl(length);
    const bitCapIntOcl inOutMask = lowMask << inOutStart;
    const bitCapIntOcl carryMask = lowMask << carryStart;

    std::unique_ptr<bitCapIntOcl[]> skipPowers(new bitCapIntOcl[controls.size() + length]);
    bitCapIntOcl controlMask = 0U;
    for (bitLenInt i = 0U; i < controls.size(); ++i) {
        bitCapIntOcl controlPower = pow2Ocl(controls[i]);
        skipPowers[i] = controlPower;
        controlMask |= controlPower;
    }
    for (bitLenInt i = 0U; i < length; ++i) {
        skipPowers[i + controls.size()] = pow2Ocl(carryStart + i);
    }
    std::sort(skipPowers.get(), skipPowers.get() + controls.size() + length);

    const bitCapIntOcl otherMask = (maxQPowerOcl - 1U) ^ (inOutMask | carryMask | controlMask);
    const bitCapIntOcl bciArgs[BCI_ARG_LEN]{ maxQPowerOcl >> ((bitLenInt)controls.size() + length), toMod,
        (bitCapIntOcl)controls.size(), controlMask, inOutMask, carryMask, otherMask, length, inOutStart, carryStart };

    const size_t sizeDiff = sizeof(bitCapIntOcl) * ((controls.size() * 2U) + length);
    AddAlloc(sizeDiff);
    BufferPtr controlBuffer = MakeBuffer(CL_MEM_COPY_HOST_PTR | CL_MEM_READ_ONLY, sizeDiff, skipPowers.get());
    skipPowers.reset();

    xMULx(api_call, bciArgs, controlBuffer);

    SubtractAlloc(sizeDiff);
}

void QEngineCUDA::CMULModx(OCLAPI api_call, bitCapIntOcl toMod, bitCapIntOcl modN, bitLenInt inOutStart,
    bitLenInt carryStart, bitLenInt length, const std::vector<bitLenInt>& controls)
{
    if (isBadBitRange(inOutStart, length, qubitCount)) {
        throw std::invalid_argument("QEngineCUDA::CMULModx range is out-of-bounds!");
    }

    if (isBadBitRange(carryStart, length, qubitCount)) {
        throw std::invalid_argument("QEngineCUDA::CMULModx range is out-of-bounds!");
    }

    ThrowIfQbIdArrayIsBad(controls, qubitCount, "QEngineCUDA::CMULModx control is out-of-bounds!");

    const bitCapIntOcl lowMask = pow2MaskOcl(length);
    const bitCapIntOcl inOutMask = lowMask << inOutStart;
    const bitCapIntOcl carryMask = lowMask << carryStart;

    std::unique_ptr<bitCapIntOcl[]> skipPowers(new bitCapIntOcl[controls.size() + length]);
    bitCapIntOcl controlMask = 0U;
    for (bitLenInt i = 0U; i < controls.size(); ++i) {
        bitCapIntOcl controlPower = pow2Ocl(controls[i]);
        skipPowers[i] = controlPower;
        controlMask |= controlPower;
    }
    for (bitLenInt i = 0U; i < length; ++i) {
        skipPowers[i + controls.size()] = pow2Ocl(carryStart + i);
    }
    std::sort(skipPowers.get(), skipPowers.get() + controls.size() + length);

    const bitCapIntOcl bciArgs[BCI_ARG_LEN]{ maxQPowerOcl, toMod, (bitCapIntOcl)controls.size(), controlMask, inOutMask,
        carryMask, modN, length, inOutStart, carryStart };

    const size_t sizeDiff = sizeof(bitCapIntOcl) * ((controls.size() * 2U) + length);
    AddAlloc(sizeDiff);
    BufferPtr controlBuffer = MakeBuffer(CL_MEM_COPY_HOST_PTR | CL_MEM_READ_ONLY, sizeDiff, skipPowers.get());
    skipPowers.reset();

    xMULx(api_call, bciArgs, controlBuffer);

    SubtractAlloc(sizeDiff);
}

/** Set 8 bit register bits based on read from classical memory */
bitCapInt QEngineCUDA::IndexedLDA(bitLenInt indexStart, bitLenInt indexLength, bitLenInt valueStart,
    bitLenInt valueLength, const unsigned char* values, bool resetValue)
{
    if (isBadBitRange(indexStart, indexLength, qubitCount)) {
        throw std::invalid_argument("QEngineCUDA::IndexedLDA range is out-of-bounds!");
    }

    if (isBadBitRange(valueStart, valueLength, qubitCount)) {
        throw std::invalid_argument("QEngineCUDA::IndexedLDA range is out-of-bounds!");
    }

    if (!stateBuffer) {
        return 0U;
    }

    if (resetValue) {
        SetReg(valueStart, valueLength, ZERO_BCI);
    }

    const bitLenInt valueBytes = (valueLength + 7) / 8;
    const bitCapIntOcl inputMask = bitRegMaskOcl(indexStart, indexLength);
    const bitCapIntOcl bciArgs[BCI_ARG_LEN]{ maxQPowerOcl >> valueLength, indexStart, inputMask, valueStart, valueBytes,
        valueLength, 0U, 0U, 0U, 0U };

    ArithmeticCall(OCL_API_INDEXEDLDA, bciArgs, values, pow2Ocl(indexLength) * valueBytes);

#if ENABLE_VM6502Q_DEBUG
    return (bitCapIntOcl)(GetExpectation(valueStart, valueLength) + (real1_f)0.5f);
#else
    return ZERO_BCI;
#endif
}

/** Add or Subtract based on an indexed load from classical memory */
bitCapIntOcl QEngineCUDA::OpIndexed(OCLAPI api_call, bitCapIntOcl carryIn, bitLenInt indexStart, bitLenInt indexLength,
    bitLenInt valueStart, bitLenInt valueLength, bitLenInt carryIndex, const unsigned char* values)
{
    if (isBadBitRange(indexStart, indexLength, qubitCount)) {
        throw std::invalid_argument("QEngineCUDA::OpIndexed range is out-of-bounds!");
    }

    if (isBadBitRange(valueStart, valueLength, qubitCount)) {
        throw std::invalid_argument("QEngineCUDA::OpIndexed range is out-of-bounds!");
    }

    if (carryIndex >= qubitCount) {
        throw std::invalid_argument("QEngineCUDA::OpIndexed carryIndex is out-of-bounds!");
    }

    if (!stateBuffer) {
        return 0U;
    }

    bool carryRes = M(carryIndex);
    // The carry has to first to be measured for its input value.
    if (carryRes) {
        /*
         * If the carry is set, we flip the carry bit. We always initially
         * clear the carry after testing for carry in.
         */
        carryIn ^= 1U;
        X(carryIndex);
    }

    const bitLenInt valueBytes = (valueLength + 7) / 8;
    const bitCapIntOcl lengthPower = pow2Ocl(valueLength);
    const bitCapIntOcl carryMask = pow2Ocl(carryIndex);
    const bitCapIntOcl inputMask = bitRegMaskOcl(indexStart, indexLength);
    const bitCapIntOcl outputMask = bitRegMaskOcl(valueStart, valueLength);
    const bitCapIntOcl otherMask = (maxQPowerOcl - 1U) & (~(inputMask | outputMask | carryMask));
    const bitCapIntOcl bciArgs[BCI_ARG_LEN]{ maxQPowerOcl >> 1U, indexStart, inputMask, valueStart, outputMask,
        otherMask, carryIn, carryMask, lengthPower, valueBytes };

    ArithmeticCall(api_call, bciArgs, values, pow2Ocl(indexLength) * valueBytes);

#if ENABLE_VM6502Q_DEBUG
    return (bitCapIntOcl)(GetExpectation(valueStart, valueLength) + (real1_f)0.5f);
#else
    return 0U;
#endif
}

/** Add based on an indexed load from classical memory */
bitCapInt QEngineCUDA::IndexedADC(bitLenInt indexStart, bitLenInt indexLength, bitLenInt valueStart,
    bitLenInt valueLength, bitLenInt carryIndex, const unsigned char* values)
{
    return OpIndexed(OCL_API_INDEXEDADC, 0U, indexStart, indexLength, valueStart, valueLength, carryIndex, values);
}

/** Subtract based on an indexed load from classical memory */
bitCapInt QEngineCUDA::IndexedSBC(bitLenInt indexStart, bitLenInt indexLength, bitLenInt valueStart,
    bitLenInt valueLength, bitLenInt carryIndex, const unsigned char* values)
{
    return OpIndexed(OCL_API_INDEXEDSBC, 1, indexStart, indexLength, valueStart, valueLength, carryIndex, values);
}

/** Set 8 bit register bits based on read from classical memory */
void QEngineCUDA::Hash(bitLenInt start, bitLenInt length, const unsigned char* values)
{
    const bitLenInt bytes = (length + 7) / 8;
    const bitCapIntOcl inputMask = bitRegMaskOcl(start, length);
    const bitCapIntOcl bciArgs[BCI_ARG_LEN]{ maxQPowerOcl, start, inputMask, bytes, 0U, 0U, 0U, 0U, 0U, 0U };

    ArithmeticCall(OCL_API_HASH, bciArgs, values, pow2Ocl(length) * bytes);
}

void QEngineCUDA::PhaseFlipX(OCLAPI api_call, const bitCapIntOcl* bciArgs)
{
    CHECK_ZERO_SKIP();

    PoolItemPtr poolItem = GetFreePoolItem();

    DISPATCH_TEMP_WRITE(poolItem->ulongBuffer, sizeof(bitCapIntOcl) * 5, bciArgs);

    const size_t ngc = FixWorkItemCount(bciArgs[0], nrmGroupCount);
    const size_t ngs = FixGroupSize(ngc, nrmGroupSize);

    hipStreamSynchronize(device_context->params_queue);

    QueueCall(api_call, ngc, ngs, { stateBuffer, poolItem->ulongBuffer });
}

void QEngineCUDA::CPhaseFlipIfLess(bitCapInt greaterPerm, bitLenInt start, bitLenInt length, bitLenInt flagIndex)
{
    if (isBadBitRange(start, length, qubitCount)) {
        throw std::invalid_argument("QEngineCUDA::CPhaseFlipIfLess range is out-of-bounds!");
    }

    if (flagIndex >= qubitCount) {
        throw std::invalid_argument("QEngineCUDA::CPhaseFlipIfLess flagIndex is out-of-bounds!");
    }

    const bitCapIntOcl bciArgs[BCI_ARG_LEN]{ maxQPowerOcl >> 1U, bitRegMaskOcl(start, length), pow2Ocl(flagIndex),
        greaterPerm.bits[0U], start, 0U, 0U, 0U, 0U, 0U };

    PhaseFlipX(OCL_API_CPHASEFLIPIFLESS, bciArgs);
}

void QEngineCUDA::PhaseFlipIfLess(bitCapInt greaterPerm, bitLenInt start, bitLenInt length)
{
    if (isBadBitRange(start, length, qubitCount)) {
        throw std::invalid_argument("QEngineCUDA::PhaseFlipIfLess range is out-of-bounds!");
    }

    const bitCapIntOcl bciArgs[BCI_ARG_LEN]{ maxQPowerOcl >> 1U, bitRegMaskOcl(start, length), greaterPerm.bits[0U],
        start, 0U, 0U, 0U, 0U, 0U, 0U };

    PhaseFlipX(OCL_API_PHASEFLIPIFLESS, bciArgs);
}
#endif

/// Set arbitrary pure quantum state, in unsigned int permutation basis
void QEngineCUDA::SetQuantumState(const complex* inputState)
{
    clDump();

    if (!stateBuffer) {
        ReinitBuffer();
    }

    DISPATCH_BLOCK_WRITE(stateBuffer, 0U, sizeof(complex) * maxQPowerOcl, inputState);

    UpdateRunningNorm();
}

bitCapInt QEngineCUDA::MAll()
{
    if (!stateBuffer) {
        return 0U;
    }

    // It's much more costly, by the end, to read amplitudes one-at-a-time from the GPU instead of all-at-once. However,
    // we might need to less work, overall, if we generate an (unbiased) sample before "walking" the full probability
    // distribution. Hence, if we try this special-case approach, we should mask GPU-read latency with non-blocking
    // calls.

    constexpr size_t cReadWidth = (QRACK_ALIGN_SIZE > sizeof(complex)) ? (QRACK_ALIGN_SIZE / sizeof(complex)) : 1U;
    const size_t alignSize = (maxQPowerOcl > cReadWidth) ? cReadWidth : maxQPowerOcl;
    const real1_f rnd = Rand();
    real1_f totProb = ZERO_R1_F;
    bitCapIntOcl lastNonzero = maxQPowerOcl - 1U;
    bitCapIntOcl perm = 0U;
    std::unique_ptr<complex[]> amp(new complex[alignSize]);
    DISPATCH_BLOCK_READ(stateBuffer, sizeof(complex) * perm, sizeof(complex) * alignSize, amp.get());
    while (perm < maxQPowerOcl) {
        Finish();
        const std::vector<complex> partAmp{ amp.get(), amp.get() + alignSize };
        if ((perm + alignSize) < maxQPowerOcl) {
            tryCuda("Failed to read buffer", [&] {
                return hipMemcpyAsync((void*)amp.get(), (void*)(((complex*)stateBuffer.get()) + perm + alignSize),
                    sizeof(complex) * alignSize, hipMemcpyDeviceToHost, device_context->queue);
            });
        }
        for (size_t i = 0U; i < alignSize; ++i) {
            const real1_f partProb = (real1_f)norm(partAmp[i]);
            if (partProb > REAL1_EPSILON) {
                totProb += partProb;
                if ((totProb > rnd) || ((ONE_R1_F - totProb) <= FP_NORM_EPSILON)) {
                    SetPermutation(perm);
                    return perm;
                }
                lastNonzero = perm;
            }
            ++perm;
        }
    }

    SetPermutation(lastNonzero);
    return lastNonzero;
}

complex QEngineCUDA::GetAmplitude(bitCapInt perm)
{
    if (bi_compare(perm, maxQPower) >= 0) {
        throw std::invalid_argument("QEngineCUDA::GetAmplitude argument out-of-bounds!");
    }

    // WARNING: Does not normalize!
    if (!stateBuffer) {
        return ZERO_CMPLX;
    }

    complex amp;
    DISPATCH_BLOCK_READ(stateBuffer, perm.bits[0U], sizeof(complex), &amp);

    return amp;
}

void QEngineCUDA::SetAmplitude(bitCapInt perm, complex amp)
{
    if (bi_compare(perm, maxQPower) >= 0) {
        throw std::invalid_argument("QEngineCUDA::SetAmplitude argument out-of-bounds!");
    }

    if (!stateBuffer && !norm(amp)) {
        return;
    }

    if (!stateBuffer) {
        ReinitBuffer();
        ClearBuffer(stateBuffer, 0U, maxQPowerOcl);
    }

    permutationAmp = amp;

    if (runningNorm != REAL1_DEFAULT_ARG) {
        runningNorm += norm(amp) - norm(permutationAmp);
    }

    tryCuda("Failed to enqueue buffer write", [&] {
        return hipMemcpy((void*)((complex*)(stateBuffer.get()) + maxQPowerOcl), (void*)&permutationAmp,
            sizeof(complex), hipMemcpyHostToDevice);
    });
}

/// Get pure quantum state, in unsigned int permutation basis
void QEngineCUDA::GetQuantumState(complex* outputState)
{
    if (doNormalize) {
        NormalizeState();
    }

    if (!stateBuffer) {
        std::fill(outputState, outputState + maxQPowerOcl, ZERO_CMPLX);
        return;
    }

    DISPATCH_BLOCK_READ(stateBuffer, 0U, sizeof(complex) * maxQPowerOcl, outputState);
}

/// Get all probabilities, in unsigned int permutation basis
void QEngineCUDA::GetProbs(real1* outputProbs) { ProbRegAll(0U, qubitCount, outputProbs); }

real1_f QEngineCUDA::SumSqrDiff(QEngineCUDAPtr toCompare)
{
    if (!toCompare) {
        return ONE_R1_F;
    }

    if (this == toCompare.get()) {
        return ZERO_R1_F;
    }

    // If the qubit counts are unequal, these can't be approximately equal objects.
    if (qubitCount != toCompare->qubitCount) {
        // Max square difference:
        return ONE_R1_F;
    }

    // Make sure both engines are normalized
    if (doNormalize) {
        NormalizeState();
    }
    if (toCompare->doNormalize) {
        toCompare->NormalizeState();
    }

    if (!stateBuffer && !toCompare->stateBuffer) {
        return ZERO_R1_F;
    }

    if (!stateBuffer) {
        toCompare->UpdateRunningNorm();
        return (real1_f)(toCompare->runningNorm);
    }

    if (!toCompare->stateBuffer) {
        UpdateRunningNorm();
        return (real1_f)runningNorm;
    }

    if (randGlobalPhase) {
        real1_f lPhaseArg = FirstNonzeroPhase();
        real1_f rPhaseArg = toCompare->FirstNonzeroPhase();
        NormalizeState(REAL1_DEFAULT_ARG, REAL1_DEFAULT_ARG, rPhaseArg - lPhaseArg);
    }

    toCompare->clFinish();

    const bitCapIntOcl bciArgs[BCI_ARG_LEN]{ maxQPowerOcl, 0U, 0U, 0U, 0U, 0U, 0U, 0U, 0U, 0U };

    PoolItemPtr poolItem = GetFreePoolItem();

    DISPATCH_WRITE(poolItem->ulongBuffer, sizeof(bitCapIntOcl), bciArgs);

    const size_t ngc = FixWorkItemCount(maxQPowerOcl, nrmGroupCount);
    const size_t ngs = FixGroupSize(ngc, nrmGroupSize);

    const size_t partInnerSize = ngc / ngs;

    AddAlloc(sizeof(complex) * partInnerSize);
    BufferPtr locCmplxBuffer = MakeBuffer(CL_MEM_READ_ONLY, sizeof(complex) * partInnerSize);

    QueueCall(OCL_API_APPROXCOMPARE, ngc, ngs,
        { stateBuffer, toCompare->stateBuffer, poolItem->ulongBuffer, locCmplxBuffer }, sizeof(complex) * ngs);

    std::unique_ptr<complex[]> partInner(new complex[partInnerSize]);

    clFinish();
    tryCuda("Failed to read buffer", [&] {
        return hipMemcpy(
            (void*)(partInner.get()), locCmplxBuffer.get(), sizeof(complex) * partInnerSize, hipMemcpyDeviceToHost);
    });
    locCmplxBuffer.reset();
    SubtractAlloc(sizeof(complex) * partInnerSize);

    complex totInner = ZERO_CMPLX;
    for (size_t i = 0; i < partInnerSize; ++i) {
        totInner += partInner[i];
    }

    return ONE_R1_F - clampProb((real1_f)norm(totInner));
}

QInterfacePtr QEngineCUDA::Clone()
{
    if (!stateBuffer) {
        return CloneEmpty();
    }

    QEngineCUDAPtr copyPtr = std::make_shared<QEngineCUDA>(qubitCount, 0U, rand_generator, ONE_CMPLX, doNormalize,
        randGlobalPhase, useHostRam, deviceID, hardware_rand_generator != NULL, false, (real1_f)amplitudeFloor);

    copyPtr->clFinish();
    clFinish();

    tryCuda("Failed to enqueue buffer copy", [&] {
        return hipMemcpy(
            copyPtr->stateBuffer.get(), stateBuffer.get(), sizeof(complex) * maxQPowerOcl, hipMemcpyDeviceToDevice);
    });

    copyPtr->runningNorm = runningNorm;

    return copyPtr;
}

QEnginePtr QEngineCUDA::CloneEmpty()
{
    QEngineCUDAPtr copyPtr = std::make_shared<QEngineCUDA>(0U, 0U, rand_generator, ONE_CMPLX, doNormalize,
        randGlobalPhase, useHostRam, deviceID, hardware_rand_generator != NULL, false, (real1_f)amplitudeFloor);

    copyPtr->SetQubitCount(qubitCount);

    return copyPtr;
}

void QEngineCUDA::NormalizeState(real1_f nrm, real1_f norm_thresh, real1_f phaseArg)
{
    CHECK_ZERO_SKIP();

    if ((runningNorm == REAL1_DEFAULT_ARG) && (nrm == REAL1_DEFAULT_ARG)) {
        UpdateRunningNorm();
    }

    if (nrm < ZERO_R1) {
        // runningNorm can be set by OpenCL queue pop, so finish first.
        clFinish();
        nrm = (real1_f)runningNorm;
    }
    // We might avoid the clFinish().
    if (nrm <= FP_NORM_EPSILON) {
        ZeroAmplitudes();
        return;
    }
    if ((abs(ONE_R1 - nrm) <= FP_NORM_EPSILON) && ((phaseArg * phaseArg) <= FP_NORM_EPSILON)) {
        return;
    }
    // We might have async execution of gates still happening.
    clFinish();

    if (norm_thresh < ZERO_R1) {
        norm_thresh = (real1_f)amplitudeFloor;
    }
    nrm = ONE_R1_F / std::sqrt((real1_s)nrm);

    PoolItemPtr poolItem = GetFreePoolItem();

    complex c_args[2]{ complex((real1)norm_thresh, ZERO_R1), std::polar((real1)nrm, (real1)phaseArg) };
    DISPATCH_TEMP_WRITE(poolItem->cmplxBuffer, sizeof(complex) * 2, c_args);

    bitCapIntOcl bciArgs[1]{ maxQPowerOcl };
    DISPATCH_TEMP_WRITE(poolItem->ulongBuffer, sizeof(bitCapIntOcl), bciArgs);

    const size_t ngc = FixWorkItemCount(maxQPowerOcl, nrmGroupCount);
    const size_t ngs = FixGroupSize(ngc, nrmGroupSize);

    OCLAPI api_call;
    if (maxQPowerOcl == ngc) {
        api_call = OCL_API_NORMALIZE_WIDE;
    } else {
        api_call = OCL_API_NORMALIZE;
    }

    QueueCall(api_call, ngc, ngs, { stateBuffer, poolItem->ulongBuffer, poolItem->cmplxBuffer });
    QueueSetRunningNorm(ONE_R1_F);
}

void QEngineCUDA::UpdateRunningNorm(real1_f norm_thresh)
{
    if (!stateBuffer) {
        runningNorm = ZERO_R1_F;
        return;
    }

    if (norm_thresh < ZERO_R1) {
        norm_thresh = (real1_f)amplitudeFloor;
    }

    PoolItemPtr poolItem = GetFreePoolItem();

    const real1 r1_args[1]{ (real1)norm_thresh };
    DISPATCH_TEMP_WRITE(poolItem->realBuffer, sizeof(real1), r1_args);
    DISPATCH_TEMP_WRITE(poolItem->ulongBuffer, sizeof(bitCapIntOcl), &maxQPowerOcl);

    const size_t ngc = FixWorkItemCount(maxQPowerOcl, nrmGroupCount);
    const size_t ngs = FixGroupSize(ngc, nrmGroupSize);

    QueueCall(OCL_API_UPDATENORM, ngc, ngs, { stateBuffer, poolItem->ulongBuffer, poolItem->realBuffer, nrmBuffer },
        sizeof(real1) * ngs);

    WAIT_REAL1_SUM(nrmBuffer, ngc / ngs, nrmArray, &runningNorm);

    if (runningNorm <= FP_NORM_EPSILON) {
        ZeroAmplitudes();
    }
}

#if defined(__APPLE__)
complex* _aligned_state_vec_alloc(bitCapIntOcl allocSize)
{
    void* toRet;
    posix_memalign(&toRet, QRACK_ALIGN_SIZE, allocSize);
    return (complex*)toRet;
}
#endif

std::shared_ptr<complex> QEngineCUDA::AllocStateVec(bitCapIntOcl elemCount, bool doForceAlloc)
{
    // If we're not using host ram, there's no reason to allocate.
    if (!elemCount || (!doForceAlloc && !stateVec)) {
        return NULL;
    }

#if defined(__ANDROID__)
    return std::shared_ptr<complex>(elemCount);
#else
    // elemCount is always a power of two, but might be smaller than QRACK_ALIGN_SIZE
    size_t allocSize = sizeof(complex) * (size_t)elemCount;
    if (allocSize < QRACK_ALIGN_SIZE) {
        allocSize = QRACK_ALIGN_SIZE;
    }
#if defined(__APPLE__)
    return std::shared_ptr<complex>(_aligned_state_vec_alloc(allocSize), [](complex* c) { free(c); });
#elif defined(_WIN32) && !defined(__CYGWIN__)
    return std::shared_ptr<complex>(
        (complex*)_aligned_malloc(allocSize, QRACK_ALIGN_SIZE), [](complex* c) { _aligned_free(c); });
#else
    return std::shared_ptr<complex>((complex*)aligned_alloc(QRACK_ALIGN_SIZE, allocSize), [](complex* c) { free(c); });
#endif
#endif
}

BufferPtr QEngineCUDA::MakeStateVecBuffer(std::shared_ptr<complex> nStateVec)
{
    if (!maxQPowerOcl) {
        return NULL;
    }

    if (nStateVec) {
        return MakeBuffer(CL_MEM_USE_HOST_PTR | CL_MEM_READ_WRITE, sizeof(complex) * maxQPowerOcl, nStateVec.get());
    } else {
        return MakeBuffer(CL_MEM_READ_WRITE, sizeof(complex) * maxQPowerOcl);
    }
}

void QEngineCUDA::ReinitBuffer()
{
    AddAlloc(sizeof(complex) * maxQPowerOcl);
    stateVec = AllocStateVec(maxQPowerOcl, usingHostRam);
    ResetStateBuffer(MakeStateVecBuffer(stateVec));
}

void QEngineCUDA::ClearBuffer(BufferPtr buff, bitCapIntOcl offset, bitCapIntOcl size)
{
    tryCuda("Failed to enqueue buffer write", [&] {
        return hipMemsetAsync(
            (void*)(((complex*)buff.get()) + offset), 0, size * sizeof(complex), device_context->queue);
    });
}

} // namespace Qrack
